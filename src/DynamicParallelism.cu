#include "hip/hip_runtime.h"
#include "DynamicParallelism.h"

__device__ double error;


template<typename T>
void DynamicParallelism::_cudaCheck(T e, const char* func, const char* call, const int line){
    if(e != hipSuccess){
        printf("\"%s\" at %d in %s\n\treturned %d\n-> %s\n", func, line, call, (int)e, hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }
}

// Divergence kernel
__global__ void divergence(double *d_u0, double *d_v0, double *d_w0, double *d_R, float *d_e, float *d_f, float *d_g, 
						float *d_h, float *d_m, float *d_n, int alpha1, int  nx, int  ny, int nz,float dx,float dy,float dz)
{

    int icell_cent = blockDim.x*blockIdx.x+threadIdx.x;
    int k = icell_cent/((nx-1)*(ny-1));
    int j = (icell_cent - k*(nx-1)*(ny-1))/(nx-1);
    int i = icell_cent - k*(nx-1)*(ny-1) - j*(nx-1);
    int icell_face = i + j*nx + k*nx*ny;

    if((i<nx-1)&&(j<ny-1)&&(k<nz-1)){
        d_R[icell_cent] = (-2*pow(alpha1, 2.0))*(((d_e[icell_cent]*d_u0[icell_face+1]-d_f[icell_cent]*d_u0[icell_face])*dx)+
						((d_g[icell_cent]*d_v0[icell_face + nx]-d_h[icell_cent]*d_v0[icell_face])*dy)+((d_m[icell_cent]*
						d_w0[icell_face + nx*ny]-d_n[icell_cent]*d_w0[icell_face])*dz));         // Divergence equation
    }
}


__global__ void SOR_RB(double *d_lambda, int nx, int ny, int nz, float omega, float  A, float  B, float  dx, float *d_e, 
						float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, double *d_R, int offset)
{
    
    int icell_cent = blockDim.x*blockIdx.x+threadIdx.x;
    int k = icell_cent/((nx-1)*(ny-1));
    int j = (icell_cent - k*(nx-1)*(ny-1))/(nx-1);
    int i = icell_cent - k*(nx-1)*(ny-1) - j*(nx-1);
    
    if ( (i > 0) && (i < nx-2) && (j > 0) && (j < ny-2) && (k < nz-2) && (k > 0) && ((i+j+k)%2) == offset ){
        
        d_lambda[icell_cent] = (omega/(d_e[icell_cent]+d_f[icell_cent]+d_g[icell_cent]+d_h[icell_cent]+d_m[icell_cent]+
								d_n[icell_cent]))*(d_e[icell_cent]*d_lambda[icell_cent+1]+d_f[icell_cent]*
								d_lambda[icell_cent-1]+d_g[icell_cent]*d_lambda[icell_cent + (nx-1)]+d_h[icell_cent]*
								d_lambda[icell_cent - (nx-1)]+d_m[icell_cent]*d_lambda[icell_cent + (nx-1)*(ny-1)]+
								d_n[icell_cent]*d_lambda[icell_cent - (nx-1)*(ny-1)]-d_R[icell_cent])+
								(1-omega)*d_lambda[icell_cent];    /// SOR formulation
    }
}

__global__ void assign_lambda_to_lambda_old(double *d_lambda, double *d_lambda_old, int nx, int ny, int nz) {
    
    int ii = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(ii < (nz-1)*(ny-1)*(nx-1)) {
        d_lambda_old[ii] = d_lambda[ii];
    }
    
}

__global__ void applyNeumannBC(double *d_lambda, int nx, int ny) {
    // Neumann boundary condition (lambda (@k=0) = lambda (@k=1))
    int ii = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(ii < nx*ny) {
      d_lambda[ii] = d_lambda[ii + 1*(nx-1)*(ny-1)];
    }
}

__global__ void calculateError(double *d_lambda, double *d_lambda_old, int nx, int ny, int nz, double *d_value, 
								double *d_bvalue)
{


    int d_size = (nx-1)*(ny-1)*(nz-1);
    int ii = blockDim.x*blockIdx.x+threadIdx.x;
    int numblocks = (d_size/BLOCKSIZE) +1;

    if (ii < d_size){
        d_value[ii] = fabs(d_lambda[ii] - d_lambda_old[ii])/((nx-1)*(ny-1)*(nz-1));
    }
    __syncthreads();
        double sum = 0.0;
    if (threadIdx.x > 0){ 
        return;
    }
    if (threadIdx.x == 0) {
         for (int j=0; j<BLOCKSIZE; j++){
        int index = blockIdx.x*blockDim.x+j;
        if (index<d_size){
            sum += d_value[index]; 
        }
         }
    }
    
    __syncthreads();
    d_bvalue[blockIdx.x] = sum;

    if (ii>0){
        return;
    }

    error = 0.0;
    if (ii==0){
        for (int k =0; k<numblocks; k++){
        error += d_bvalue[k];
        }
    }

 }

// Euler Final Velocity kernel
__global__ void finalVelocity(double *d_u0, double *d_v0, double *d_w0, double *d_lambda, double *d_u, double *d_v,
							 double *d_w, int *d_icellflag, float *d_f, float *d_h, float *d_n, int alpha1, int alpha2, 
							 float dx, float dy, float dz, int  nx, int  ny, int nz)
{

    int icell_face = blockDim.x*blockIdx.x+threadIdx.x;
    int k = icell_face/(nx*ny);
    int j = (icell_face - k*nx*ny)/nx;
    int i = icell_face - k*nx*ny - j*nx;
    int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values

    if((i<nx)&&(j<ny)&&(k<nz)){
        
        d_u[icell_face] = d_u0[icell_face];
        d_v[icell_face] = d_v0[icell_face];
        d_w[icell_face] = d_w0[icell_face];

    }

    
    if ((i > 0) && (i < nx-1) && (j > 0) && (j < ny-1) && (k < nz-1) && (k > 0)) {

        d_u[icell_face] = d_u0[icell_face]+(1/(2*pow(alpha1, 2.0)*dx))*d_f[icell_cent]*dx*dx*
						 (d_lambda[icell_cent]-d_lambda[icell_cent-1]);
        d_v[icell_face] = d_v0[icell_face]+(1/(2*pow(alpha1, 2.0)*dy))*d_h[icell_cent]*dy*dy*
						 (d_lambda[icell_cent]-d_lambda[icell_cent - (nx-1)]);
        d_w[icell_face] = d_w0[icell_face]+(1/(2*pow(alpha2, 2.0)*dz))*d_n[icell_cent]*dz*dz*
						 (d_lambda[icell_cent]-d_lambda[icell_cent - (nx-1)*(ny-1)]);

    }

    
    if ((i > 0) && (i < nx-1) && (j > 0) && (j < ny-1) && (k < nz-1) && (k > 0) && (d_icellflag[icell_cent] == 0) ) {
        d_u[icell_face] = 0;
        d_u[icell_face+1] = 0;
        d_v[icell_face] = 0;
        d_v[icell_face+nx] = 0;
        d_w[icell_face] = 0;
        d_w[icell_face+nx*ny] = 0;
        
    }
}
__global__ void SOR_iteration (double *d_lambda, double *d_lambda_old, int nx, int ny, int nz, float omega, float  A, 
								float  B, float  dx, float *d_e, float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, 
								double *d_R, int itermax, double tol, double *d_value, double *d_bvalue, double *d_u0, 
								double *d_v0, double *d_w0,int alpha1, int alpha2, float dy, float dz, double *d_u, 
								double *d_v, double *d_w, int *d_icellflag) 
{
    int iter = 0;
    error = 1.0;

    // Calculate divergence of initial velocity field
    dim3 numberOfThreadsPerBlock(BLOCKSIZE,1,1);
    dim3 numberOfBlocks(ceil(((nx-1)*(ny-1)*(nz-1))/(double) (BLOCKSIZE)),1,1);
    // Invoke divergence kernel
    divergence<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_u0,d_v0,d_w0,d_R,d_e,d_f,d_g,d_h,d_m,d_n,alpha1,nx,ny,nz,dx,dy,
															dz);

    // Iterate untill convergence is reached
    while ( (iter < itermax) && (error > tol)) {
        
        // Save previous iteration values for error calculation 
        assign_lambda_to_lambda_old<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, nx, ny, nz);
        hipDeviceSynchronize();
        // SOR part
        int offset = 0;   // red nodes
        // Invoke red-black SOR kernel for red nodes
        SOR_RB<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda, nx, ny, nz, omega, A, B, dx, d_e, d_f, d_g, d_h, d_m, 
															d_n, d_R, offset);
        hipDeviceSynchronize();
        offset = 1;    // black nodes
        // Invoke red-black SOR kernel for black nodes
        SOR_RB<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda, nx, ny, nz, omega, A, B, dx, d_e, d_f, d_g, d_h, d_m, 
															d_n, d_R,offset);
        hipDeviceSynchronize();
        dim3 numberOfBlocks2(ceil(((nx-1)*(ny-1))/(double) (BLOCKSIZE)),1,1);
        // Invoke kernel to apply Neumann boundary condition (lambda (@k=0) = lambda (@k=1))
        applyNeumannBC<<<numberOfBlocks2,numberOfThreadsPerBlock>>>(d_lambda, nx, ny);
        hipDeviceSynchronize();
        // Error calculation
        calculateError<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda,d_lambda_old, nx, ny, nz, d_value,d_bvalue);
        hipDeviceSynchronize();

        iter += 1;
        
    }
    printf("number of iteration = %d\n", iter);
    printf("error = %2.9f\n", error);
    dim3 numberOfBlocks3(ceil((nx*ny*nz)/(double) (BLOCKSIZE)),1,1);
    // Invoke final velocity (Euler) kernel
    finalVelocity<<<numberOfBlocks3,numberOfThreadsPerBlock>>>(d_u0,d_v0,d_w0,d_lambda,d_u,d_v,d_w,d_icellflag,d_f,d_h,d_n,
																alpha1,alpha2,dx,dy,dz,nx,ny,nz);
}



void DynamicParallelism::solve(bool solveWind) 
{
<<<<<<< HEAD
    auto startTotal = std::chrono::high_resolution_clock::now(); // Start recording execution time    


    long numcell_cent = (nx-1)*(ny-1)*(nz-1);         /// Total number of cell-centered values in domain
    long numface_cent = nx*ny*nz;                     /// Total number of face-centered values in domain
    

    // Declare coefficients for SOR solver
    float *e, *f, *g, *h, *m, *n;
    e = new float [numcell_cent];
    f = new float [numcell_cent];
    g = new float [numcell_cent];
    h = new float [numcell_cent];
    m = new float [numcell_cent];
    n = new float [numcell_cent];

    float *d_e, *d_f, *d_g, *d_h, *d_m, *d_n;
=======
    
	int numblocks = (numcell_cent/BLOCKSIZE)+1;

    std::vector<double> value(numcell_cent,0.0);
    std::vector<double> bvalue(numblocks,0.0);   
	double *d_u0, *d_v0, *d_w0; 
	double *d_value,*d_bvalue;
    float *d_x,*d_y,*d_z;
    double *d_u, *d_v, *d_w;  
	int *d_icellflag;


    auto start = std::chrono::high_resolution_clock::now(); // Start recording execution time

>>>>>>> 1d5d8aa6c846d4cc653130b2767ebd5338e81607
    hipMalloc((void **) &d_e, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_f, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_g, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_h, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_m, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_n, numcell_cent * sizeof(float));
<<<<<<< HEAD

    /// Declaration of initial wind components (u0,v0,w0)
    double *u0, *v0, *w0;
    u0 = new double [numface_cent];
    v0 = new double [numface_cent];
    w0 = new double [numface_cent];
    
    
    double *R, *d_R;              //!> Divergence of initial velocity field
    R = new double [numcell_cent];
    hipMalloc((void **) &d_R, numcell_cent * sizeof(double));    

    // Declare Lagrange multipliers
    /// Declaration of Lagrange multipliers
    double *lambda, *lambda_old, *d_lambda, *d_lambda_old;
    lambda = new double [numcell_cent];
    lambda_old = new double [numcell_cent];
    hipMalloc((void **) &d_lambda, numcell_cent * sizeof(double));
    hipMalloc((void **) &d_lambda_old, numcell_cent * sizeof(double));

    int numblocks = (numcell_cent/BLOCKSIZE)+1;
    double *value, *bvalue;
    value = new double [numcell_cent];
    bvalue = new double [numblocks];    
    
    
    for ( int i = 0; i < nx-1; i++){
        x.push_back((i+0.5)*dx);         /// Location of face centers in x-dir
    }
    for ( int j = 0; j < ny-1; j++){
        y.push_back( (j+0.5)*dy);         /// Location of face centers in y-dir
    }
    for ( int k = 0; k < nz-1; k++){
        z.push_back( (k-0.5)*dz);         /// Location of face centers in z-dir
    }


    /*
    Set Terrain buildings
    Deprecate
    */
    if (mesh)
    {
        std::cout << "Creating terrain blocks...\n";
        for (int i = 0; i < nx; i++)
        {
            for (int j = 0; j < ny; j++)
            {           //get height, then add half a cell, if the height exceeds half of a cell partially, it will round up.
                float heightToMesh = mesh->getHeight(i * dx + dx * 0.5f, j * dy + dy * 0.5f) + 0.5f * dz;
                for (int k = 0; k < (int)(heightToMesh / dz); k++)
                    buildings.push_back(new RectangularBuilding(i * dx, j * dy, k * dz, dx, dy, dz));
            }
             printProgress( (float)i / (float)nx);
        }
        std::cout << "blocks created\n";
    }


    float z0 = 0.1;                 /// Surface roughness
    float z_ref = 10.0;             /// Height of the measuring sensor (m)
    float U_ref = 5.0;              /// Measured velocity at the sensor height (m/s)
    /*//float H = 20.0;                 /// Building height
    //float W = 20.0;                 /// Building width
    //float L = 20.0;                 /// Building length
    float x_start = 90.0;           /// Building start location in x-direction
    float y_start = 90.0;           /// Building start location in y-direction
    float i_start = std::round(x_start/dx);     /// Index of building start location in x-direction
    float i_end = std::round((x_start+20.0)/dx);   /// Index of building end location in x-direction
    float j_start = std::round(y_start/dy);     /// Index of building start location in y-direction
    float j_end = std::round((y_start+20.0)/dy);   /// Index of building end location in y-direction 
    float k_end = std::round(20.0/dz);             /// Index of building end location in z-direction*/
    int *icellflag, *d_icellflag;
    icellflag = new int [numcell_cent];       /// Cell index flag (0 = building, 1 = fluid)

    /*std::cout << "i_start:" << i_start << "\n";   // Print the number of iterations
    std::cout << "i_end:" << i_end << "\n";       // Print the number of iterations
    std::cout << "j_start:" << j_start << "\n";   // Print the number of iterations
    std::cout << "j_end:" << j_end << "\n";       // Print the number of iterations    
    std::cout << "k_end:" << k_end << "\n";       // Print the number of iterations */

    for ( int k = 0; k < nz-1; k++){
        for (int j = 0; j < ny-1; j++){
            for (int i = 0; i < nx-1; i++){

                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);            /// Lineralized index for cell centered values
                e[icell_cent] = f[icell_cent] = g[icell_cent] = h[icell_cent] = m[icell_cent] = n[icell_cent] = 1.0;  /// Assign initial values to the coefficients for SOR solver
                icellflag[icell_cent] = 1;                                  /// Initialize all cells to fluid   
                lambda[icell_cent] = lambda_old[icell_cent] = 0.0;
            }
        }    
    }   

    for ( int k = 1; k < nz; k++){
        for (int j = 0; j < ny; j++){
            for (int i = 0; i < nx; i++){
                
                int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values                                
                /// Define logarithmic wind profile
                u0[icell_face] = U_ref*(log((z[k]+z0)/z0)/log((z_ref+z0)/z0));
                v0[icell_face] = w0 [icell_face] = 0.0;         /// Perpendicular wind direction

            }
        }
    }

  /*  for (int k = 0; k < k_end+1; k++){
        for (int j = j_start; j < j_end; j++){
            for (int i = i_start; i < i_end; i++){

                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
                icellflag[icell_cent] = 0;                         /// Set cell index flag to building

            }
        }
    }*/

    float* zm;
    zm = new float[nz];
    int* iBuildFlag;
    iBuildFlag = new int[nx*ny*nz];
    for (int i = 0; i < buildings.size(); i++)
    {
        ((RectangularBuilding*)buildings[i])->setBoundaries(dx, dy, dz, nz, zm);
        ((RectangularBuilding*)buildings[i])->setCells(nx, ny, nz, icellflag, iBuildFlag, i);
    }

    for (int j = 0; j < ny-1; j++){
        for (int i = 0; i < nx-1; i++){
            int icell_cent = i + j*(nx-1);   /// Lineralized index for cell centered values
            icellflag[icell_cent] = 0.0;
        }
    }

   
    for (int k = 0; k < nz; k++){
        for (int j = 0; j < ny; j++){
            for (int i = 0; i < nx; i++){
                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
                int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values 
                if (icellflag[icell_cent] == 0) {
                    u0[icell_face] = 0.0;                    /// Set velocity inside the building to zero
                    u0[icell_face+1] = 0.0;
                    v0[icell_face] = 0.0;                    /// Set velocity inside the building to zero
                    v0[icell_face+nx] = 0.0;
                    w0[icell_face] = 0.0;                    /// Set velocity inside the building to zero
                    w0[icell_face+nx*ny] = 0.0;
                }
            }
        }
    }



    hipMalloc((void **) &d_icellflag, numcell_cent * sizeof(int));
    hipMemcpy(d_icellflag,icellflag,numcell_cent*sizeof(int),hipMemcpyHostToDevice);
    // Allocate GPU memory
    double *d_u0, *d_v0, *d_w0;
    hipMalloc((void **) &d_u0,numface_cent*sizeof(double));
    hipMalloc((void **) &d_v0,numface_cent*sizeof(double));
    hipMalloc((void **) &d_w0,numface_cent*sizeof(double));
    // Initialize GPU input/output
    hipMemcpy(d_u0,u0,numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_v0,v0,numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_w0,w0,numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_R,R,numcell_cent*sizeof(double),hipMemcpyHostToDevice);

    /// Boundary condition for building edges
    for (int k = 1; k < nz-2; k++){
        for (int j = 1; j < ny-2; j++){
            for (int i = 1; i < nx-2; i++){
                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
                if (icellflag[icell_cent] != 0) {
                    
                    /// Wall bellow
                    if (icellflag[icell_cent-(nx-1)*(ny-1)]==0) {
                        n[icell_cent] = 0.0; 

                    }
                    /// Wall above
                    if (icellflag[icell_cent+(nx-1)*(ny-1)]==0) {
                        m[icell_cent] = 0.0;
                    }
                    /// Wall in back
                    if (icellflag[icell_cent-1]==0){
                        f[icell_cent] = 0.0; 
                    }
                    /// Wall in front
                    if (icellflag[icell_cent+1]==0){
                        e[icell_cent] = 0.0; 
                    }
                    /// Wall on right
                    if (icellflag[icell_cent-(nx-1)]==0){
                        h[icell_cent] = 0.0;
                    }
                    /// Wall on left
                    if (icellflag[icell_cent+(nx-1)]==0){
                        g[icell_cent] = 0.0; 
                    }
                }
            }
        }
    }

    /// New boundary condition implementation
    for (int k = 1; k < nz-1; k++){
        for (int j = 0; j < ny-1; j++){
            for (int i = 0; i < nx-1; i++){
                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
                e[icell_cent] = e[icell_cent]/(dx*dx);
                f[icell_cent] = f[icell_cent]/(dx*dx);
                g[icell_cent] = g[icell_cent]/(dy*dy);
                h[icell_cent] = h[icell_cent]/(dy*dy);
                m[icell_cent] = m[icell_cent]/(dz*dz);
                n[icell_cent] = n[icell_cent]/(dz*dz);
            }
        }
    }

    auto startSolve = std::chrono::high_resolution_clock::now();    
    
    double *d_value,*d_bvalue;
    float *d_x,*d_y,*d_z;
=======
	hipMalloc((void **) &d_R, numcell_cent * sizeof(double)); 
    hipMalloc((void **) &d_lambda, numcell_cent * sizeof(double));
    hipMalloc((void **) &d_lambda_old, numcell_cent * sizeof(double));
	hipMalloc((void **) &d_icellflag, numcell_cent * sizeof(int));
    hipMalloc((void **) &d_u0,numface_cent*sizeof(double));
    hipMalloc((void **) &d_v0,numface_cent*sizeof(double));
    hipMalloc((void **) &d_w0,numface_cent*sizeof(double));
>>>>>>> 1d5d8aa6c846d4cc653130b2767ebd5338e81607
    hipMalloc((void **) &d_value,numcell_cent*sizeof(double));
    hipMalloc((void **) &d_bvalue,numblocks*sizeof(double));
    hipMalloc((void **) &d_x,nx*sizeof(float));
    hipMalloc((void **) &d_y,ny*sizeof(float));
    hipMalloc((void **) &d_z,nz*sizeof(float));
<<<<<<< HEAD
    hipMemcpy(d_value , value , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_bvalue , bvalue , numblocks * sizeof(double) , hipMemcpyHostToDevice);      
    hipMemcpy(d_e , e , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_f , f , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_g , g , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_h , h , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_m , m , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_n , n , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_x , x.data() , nx * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_y , y.data() , ny * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_z , z.data() , nz * sizeof(float) , hipMemcpyHostToDevice);
    
    double *d_u, *d_v, *d_w;
=======
>>>>>>> 1d5d8aa6c846d4cc653130b2767ebd5338e81607
    hipMalloc((void **) &d_u,numface_cent*sizeof(double));
    hipMalloc((void **) &d_v,numface_cent*sizeof(double));
    hipMalloc((void **) &d_w,numface_cent*sizeof(double));
	 

    hipMemcpy(d_icellflag,icellflag.data(),numcell_cent*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_u0,u0.data(),numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_v0,v0.data(),numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_w0,w0.data(),numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_R,R.data(),numcell_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_value , value.data() , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_bvalue , bvalue.data() , numblocks * sizeof(double) , hipMemcpyHostToDevice);      
    hipMemcpy(d_e , e.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_f , f.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_g , g.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_h , h.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_m , m.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_n , n.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_x , x.data() , nx * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_y , y.data() , ny * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_z , z.data() , nz * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_lambda , lambda.data() , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_lambda_old , lambda_old.data() , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);    





    /////////////////////////////////////////////////
    //                 SOR solver              //////
    /////////////////////////////////////////////////
    

    // Invoke the main (mother) kernel
    SOR_iteration<<<1,1>>>(d_lambda,d_lambda_old, nx, ny, nz, omega, A, B, dx, d_e, d_f, d_g, d_h, d_m, d_n, d_R,itermax,tol,d_value,d_bvalue,d_u0,d_v0,d_w0,alpha1,alpha2,dy,dz,d_u,d_v,d_w,d_icellflag);
    cudaCheck(hipGetLastError()); 
    
<<<<<<< HEAD
    hipMemcpy (lambda , d_lambda , numcell_cent * sizeof(double) , hipMemcpyDeviceToHost);
    hipMemcpy(u.data(), d_u,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(v.data(), d_v,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(w.data(), d_w,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
=======
    hipMemcpy (lambda.data() , d_lambda , numcell_cent * sizeof(double) , hipMemcpyDeviceToHost);
    hipMemcpy(u.data(),d_u,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(v.data(),d_v,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(w.data(),d_w,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
>>>>>>> 1d5d8aa6c846d4cc653130b2767ebd5338e81607

    hipFree (d_lambda);
    hipFree (d_e);
    hipFree (d_f);
    hipFree (d_g);
    hipFree (d_h);
    hipFree (d_m);
    hipFree (d_n);
    hipFree (d_R);
    hipFree (d_value);
    hipFree (d_bvalue);
    hipFree (d_u0);
    hipFree (d_v0);
    hipFree (d_w0);
    hipFree (d_u);
    hipFree (d_v);
    hipFree (d_w);
    hipFree (d_x);
    hipFree (d_y);
    hipFree (d_z);
    hipFree (d_icellflag);

    auto finish = std::chrono::high_resolution_clock::now();  // Finish recording execution time
<<<<<<< HEAD
    std::chrono::duration<float> elapsedTotal = finish - startTotal;
    std::chrono::duration<float> elapsedSolve = finish - startSolve;
    std::cout << "Elapsed total time: " << elapsedTotal.count() << " s\n";   // Print out elapsed execution time
    std::cout << "Elapsed solve time: " << elapsedSolve.count() << " s\n";   // Print out elapsed execution time   


}

void DynamicParallelism::outputDataFile()
{
=======
    std::chrono::duration<double> elapsed = finish - start;
    std::cout << "Elapsed time: " << elapsed.count() << " s\n";   // Print out elapsed execution time    
    

>>>>>>> 1d5d8aa6c846d4cc653130b2767ebd5338e81607
    /// Declare cell center positions
    float *x_out, *y_out, *z_out;
    x_out = new float [nx-1];
    y_out = new float [ny-1];
    z_out = new float [nz-1];


    for ( int i = 0; i < nx-1; i++) {
        x_out[i] = (i+0.5)*dx;         /// Location of cell centers in x-dir
    }
    for ( int j = 0; j < ny-1; j++){
        y_out[j] = (j+0.5)*dy;         /// Location of cell centers in y-dir
    }
    for ( int k = 0; k < nz-1; k++){
        z_out[k] = (k-0.5)*dz;         /// Location of cell centers in z-dir
    }

    	for (int k = 0; k < nz-1; k++){
            for (int j = 0; j < ny-1; j++){
                for (int i = 0; i < nx-1; i++){
                    int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values 
					int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1); 
    				u_out[icell_cent] = 0.5*(u[icell_face+1]+u[icell_face]);
    				v_out[icell_cent] = 0.5*(v[icell_face+nx]+v[icell_face]);
    				w_out[icell_cent] = 0.5*(w[icell_face+nx*ny]+w[icell_face]);
    			}
    		}	
    	}
        // Write data to file
        ofstream outdata1;
        outdata1.open("Final velocity.dat");
        if( !outdata1 ) {                 // File couldn't be opened
            cerr << "Error: file could not be opened" << endl;
            exit(1);
        }
        // Write data to file
        for (int k = 0; k < nz-1; k++){
            for (int j = 0; j < ny-1; j++){
                for (int i = 0; i < nx-1; i++){
    				int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
    				int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values
                    outdata1 << "\t" << i << "\t" << j << "\t" << k << "\t \t"<< x[i] << "\t \t" << y[j] << "\t \t" << z[k] 
							 << "\t \t"<< "\t \t" << u[icell_face] <<"\t \t"<< "\t \t"<<v[icell_face]<<"\t \t"<< "\t \t"
							 << w[icell_face]<< "\t \t"<< "\t \t" << u0[icell_face] <<"\t \t"<< "\t \t"<<v0[icell_face]
							 <<"\t \t"<< "\t \t"<<w0[icell_face]<<"\t \t"<<R[icell_cent]<< endl;   
                }
            }
        }
        outdata1.close();

<<<<<<< HEAD
    for (int k = 0; k < nz-1; k++){
        for (int j = 0; j < ny-1; j++){
            for (int i = 0; i < nx-1; i++){
                int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values 
                u_out[i][j][k] = 0.5*(u[icell_face+1]+u[icell_face]);
                v_out[i][j][k] = 0.5*(v[icell_face+nx]+v[icell_face]);
                w_out[i][j][k] = 0.5*(w[icell_face+nx*ny]+w[icell_face]);
=======

        // Write data to file
        ofstream outdata2;
        outdata2.open("Final velocity1.dat");
        if( !outdata2 ) {                 // File couldn't be opened
            cerr << "Error: file could not be opened" << endl;
            exit(1);
        }
        // Write data to file
        for (int k = 0; k < nz-1; k++){
            for (int j = 0; j < ny-1; j++){
                for (int i = 0; i < nx-1; i++){
    				int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
    				int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values
                    outdata2 << "\t" << i << "\t" << j << "\t" << k << "\t \t"<< x[i] << "\t \t" << y[j] << "\t \t" << z[k] 
							 << "\t \t"<< "\t \t" << f[icell_cent] <<"\t \t"<< "\t \t"<<e[icell_cent]<<"\t \t"<< "\t \t"
							 <<h[icell_cent]<< "\t \t"<< "\t \t" << g[icell_cent] <<"\t \t"<< "\t \t"<<n[icell_cent]<<"\t \t"
							 << "\t \t"<<m[icell_cent]<<"\t \t"<<icellflag[icell_cent]<< endl;   
                }
>>>>>>> 1d5d8aa6c846d4cc653130b2767ebd5338e81607
            }
        }
        outdata2.close(); 

<<<<<<< HEAD
    // Write data to file
    ofstream outdata;
    outdata.open("Final velocity, cell-centered.dat");
    if( !outdata ) {                 // File couldn't be opened
        cerr << "Error: file could not be opened" << endl;
        exit(1);
    }
    // Write data to file
    for (int k = 0; k < nz-1; k++){
        for (int j = 0; j < ny-1; j++){
            for (int i = 0; i < nx-1; i++){
                outdata << "\t" << i << "\t" << j << "\t" << k << "\t"<< x_out[i] << "\t" << y_out[j] << "\t" << z_out[k] << "\t" << u_out[i][j][k] << "\t" << v_out[i][j][k] << "\t" << w_out[i][j][k]                     << endl;   
            }
=======
        netcdfDat->getData(x.data(),y.data(),z.data(),u_out.data(),v_out.data(),w_out.data(),nx,ny,nz);
        //netcdfDat->getDataICell(icellflag.data(), x_out, y_out, z_out, nx-1, ny - 1, nz - 1, numcell_cent);
        //if (DTEHF)
            //netcdfDat->getCutCellFlags(cells);


       // Write data to file
/*       ofstream outdata;
        outdata.open("Final velocity, cell-centered.dat");
        if( !outdata ) {                 // File couldn't be opened
            cerr << "Error: file could not be opened" << endl;
            exit(1);
>>>>>>> 1d5d8aa6c846d4cc653130b2767ebd5338e81607
        }
        // Write data to file
        for (int k = 0; k < nz-1; k++){
            for (int j = 0; j < ny-1; j++){
                for (int i = 0; i < nx-1; i++){
					int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);
                    outdata << "\t" << i << "\t" << j << "\t" << k << "\t"<< x_out[i] << "\t" << y_out[j] << "\t" <<z_out[k] 
							<< "\t" << u_out[icell_cent] << "\t" << v_out[icell_cent] << "\t" << w_out[icell_cent]<< endl;   
    			}
    		}
    	}
        outdata.close();*/

        //netcdfDat->getDataICell(icellflag.data(), x_out, y_out, z_out, nx-1, ny - 1, nz - 1, numcell_cent);
}
    

void DynamicParallelism::outputNetCDF(  NetCDFData* netcdfDat )
{
    netcdfDat->getData(x.data(),y.data(),z.data(), u.data(), v.data(), w.data(), nx,ny,nz);
}