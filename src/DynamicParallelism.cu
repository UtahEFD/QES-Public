#include "hip/hip_runtime.h"
#include "DynamicParallelism.h"

__device__ double error;


template<typename T>
void DynamicParallelism::_cudaCheck(T e, const char* func, const char* call, const int line){
    if(e != hipSuccess){
        printf("\"%s\" at %d in %s\n\treturned %d\n-> %s\n", func, line, call, (int)e, hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }
}

// Divergence kernel
<<<<<<< HEAD
__global__ void divergence(double *d_u0, double *d_v0, double *d_w0, double *d_R, float *d_e, float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, int alpha1, int  nx, int  ny, int nz, float dx,float dy, float dz){
=======
__global__ void divergence(double *d_u0, double *d_v0, double *d_w0, double *d_R, int alpha1, int  nx, int  ny, int nz, float dx,float dy, float dz){
>>>>>>> origin/doxygenAdd

    int icell_cent = blockDim.x*blockIdx.x+threadIdx.x;
    int k = icell_cent/((nx-1)*(ny-1));
    int j = (icell_cent - k*(nx-1)*(ny-1))/(nx-1);
    int i = icell_cent - k*(nx-1)*(ny-1) - j*(nx-1);
    int icell_face = i + j*nx + k*nx*ny;

    if((i<nx-1)&&(j<ny-1)&&(k<nz-1)){
<<<<<<< HEAD
        d_R[icell_cent] = (-2*pow(alpha1, 2.0))*(((d_e[icell_cent]*d_u0[icell_face+1]-d_f[icell_cent]*d_u0[icell_face])/dx)+((d_g[icell_cent]*d_v0[icell_face + nx]-d_h[icell_cent]*d_v0[icell_face])/dy)+((d_m[icell_cent]*d_w0[icell_face + nx*ny]-d_n[icell_cent]*d_w0[icell_face])/dy));   // Divergence equation
=======
        d_R[icell_cent] = (-2*pow(alpha1, 2.0))*(((d_u0[icell_face+1]-d_u0[icell_face])/dx)+((d_v0[icell_face + nx]-d_v0[icell_face])/dy)+((d_w0[icell_face + nx*ny]-d_w0[icell_face])/dy));   // Divergence equation
>>>>>>> origin/doxygenAdd
    }
}


__global__ void SOR_RB(double *d_lambda, int nx, int ny, int nz, float omega, float  A, float  B, float  dx, float *d_e, float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, double *d_R, int offset){
    
    int icell_cent = blockDim.x*blockIdx.x+threadIdx.x;
    int k = icell_cent/((nx-1)*(ny-1));
    int j = (icell_cent - k*(nx-1)*(ny-1))/(nx-1);
    int i = icell_cent - k*(nx-1)*(ny-1) - j*(nx-1);
    
    if ( (i > 0) && (i < nx-2) && (j > 0) && (j < ny-2) && (k < nz-2) && (k > 0) && ((i+j+k)%2) == offset ){
        
        d_lambda[icell_cent] = (omega/(d_e[icell_cent]+d_f[icell_cent]+d_g[icell_cent]+d_h[icell_cent]+d_m[icell_cent]+d_n[icell_cent]))*(d_e[icell_cent]*d_lambda[icell_cent+1]+d_f[icell_cent]*d_lambda[icell_cent-1]+d_g[icell_cent]*d_lambda[icell_cent + (nx-1)]+d_h[icell_cent]*d_lambda[icell_cent - (nx-1)]+d_m[icell_cent]*d_lambda[icell_cent + (nx-1)*(ny-1)]+d_n[icell_cent]*d_lambda[icell_cent - (nx-1)*(ny-1)]-d_R[icell_cent])+(1-omega)*d_lambda[icell_cent];    /// SOR formulation
    }
}

__global__ void assign_lambda_to_lambda_old(double *d_lambda, double *d_lambda_old, int nx, int ny, int nz) {
    
    int ii = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(ii < (nz-1)*(ny-1)*(nx-1)) {
        d_lambda_old[ii] = d_lambda[ii];
    }
    
}

__global__ void applyNeumannBC(double *d_lambda, int nx, int ny) {
    // Neumann boundary condition (lambda (@k=0) = lambda (@k=1))
    int ii = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(ii < nx*ny) {
      d_lambda[ii] = d_lambda[ii + 1*(nx-1)*(ny-1)];
    }
}

__global__ void calculateError(double *d_lambda, double *d_lambda_old, int nx, int ny, int nz, double *d_value, double *d_bvalue){


    int d_size = (nx-1)*(ny-1)*(nz-1);
    int ii = blockDim.x*blockIdx.x+threadIdx.x;
    int numblocks = (d_size/BLOCKSIZE) +1;

    if (ii < d_size){
        d_value[ii] = fabs(d_lambda[ii] - d_lambda_old[ii])/((nx-1)*(ny-1)*(nz-1));
    }
    __syncthreads();
        double sum = 0.0;
    if (threadIdx.x > 0){ 
        return;
    }
    if (threadIdx.x == 0) {
         for (int j=0; j<BLOCKSIZE; j++){
        int index = blockIdx.x*blockDim.x+j;
        if (index<d_size){
            sum += d_value[index]; 
        }
         }
    }
    
    __syncthreads();
    d_bvalue[blockIdx.x] = sum;

    if (ii>0){
        return;
    }

    error = 0.0;
    if (ii==0){
        for (int k =0; k<numblocks; k++){
        error += d_bvalue[k];
        }
    }

 }

// Euler Final Velocity kernel
<<<<<<< HEAD
__global__ void finalVelocity(double *d_u0, double *d_v0, double *d_w0, double *d_lambda, double *d_u, double *d_v,double *d_w, int *d_icellflag, float *d_f, float *d_h, float *d_n, int alpha1, int alpha2, float dx, float dy, float dz, int  nx, int  ny, int nz){
=======
__global__ void finalVelocity(double *d_u0, double *d_v0, double *d_w0, double *d_lambda, double *d_u, double *d_v,double *d_w, int *d_icellflag, int alpha1, int alpha2, float dx, float dy, float dz, int  nx, int  ny, int nz){
>>>>>>> origin/doxygenAdd

    int icell_face = blockDim.x*blockIdx.x+threadIdx.x;
    int k = icell_face/(nx*ny);
    int j = (icell_face - k*nx*ny)/nx;
    int i = icell_face - k*nx*ny - j*nx;
    int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values

    if((i<nx)&&(j<ny)&&(k<nz)){
        
        d_u[icell_face] = d_u0[icell_face];
        d_v[icell_face] = d_v0[icell_face];
        d_w[icell_face] = d_w0[icell_face];

    }

    
    if ((i > 0) && (i < nx-1) && (j > 0) && (j < ny-1) && (k < nz-1) && (k > 0)) {

<<<<<<< HEAD
        d_u[icell_face] = d_u0[icell_face]+(1/(2*pow(alpha1, 2.0)*dx))*d_f[icell_cent]*(d_lambda[icell_cent]-d_lambda[icell_cent-1]);
        d_v[icell_face] = d_v0[icell_face]+(1/(2*pow(alpha1, 2.0)*dy))*d_h[icell_cent]*(d_lambda[icell_cent]-d_lambda[icell_cent - (nx-1)]);
        d_w[icell_face] = d_w0[icell_face]+(1/(2*pow(alpha2, 2.0)*dz))*d_n[icell_cent]*(d_lambda[icell_cent]-d_lambda[icell_cent - (nx-1)*(ny-1)]);
=======
        d_u[icell_face] = d_u0[icell_face]+(1/(2*pow(alpha1, 2.0)*dx))*(d_lambda[icell_cent]-d_lambda[icell_cent-1]);
        d_v[icell_face] = d_v0[icell_face]+(1/(2*pow(alpha1, 2.0)*dy))*(d_lambda[icell_cent]-d_lambda[icell_cent - (nx-1)]);
        d_w[icell_face] = d_w0[icell_face]+(1/(2*pow(alpha2, 2.0)*dz))*(d_lambda[icell_cent]-d_lambda[icell_cent - (nx-1)*(ny-1)]);
>>>>>>> origin/doxygenAdd

    }

    
    if ((i > 0) && (i < nx-1) && (j > 0) && (j < ny-1) && (k < nz-1) && (k > 0) && (d_icellflag[icell_cent] == 0) ) {
        d_u[icell_face] = 0;
        d_u[icell_face+1] = 0;
        d_v[icell_face] = 0;
        d_v[icell_face+nx] = 0;
        d_w[icell_face] = 0;
        d_w[icell_face+nx*ny] = 0;
        
    }
}
__global__ void SOR_iteration (double *d_lambda, double *d_lambda_old, int nx, int ny, int nz, float omega, float  A, float  B, float  dx, float *d_e, float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, double *d_R, int itermax, double tol, double *d_value, double *d_bvalue, double *d_u0, double *d_v0, double *d_w0,int alpha1, int alpha2, float dy, float dz, double *d_u, double *d_v, double *d_w, int *d_icellflag) {
    int iter = 0;
    error = 1.0;

    // Calculate divergence of initial velocity field
    dim3 numberOfThreadsPerBlock(BLOCKSIZE,1,1);
    dim3 numberOfBlocks(ceil(((nx-1)*(ny-1)*(nz-1))/(double) (BLOCKSIZE)),1,1);
    // Invoke divergence kernel
<<<<<<< HEAD
    divergence<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_u0,d_v0,d_w0,d_R,d_e,d_f,d_g,d_h,d_m,d_n,alpha1,nx,ny,nz,dx,dy,dz);
=======
    divergence<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_u0,d_v0,d_w0,d_R,alpha1,nx,ny,nz,dx,dy,dz);
>>>>>>> origin/doxygenAdd

    // Iterate untill convergence is reached
    while ( (iter < itermax) && (error > tol)) {
        
        // Save previous iteration values for error calculation 
        assign_lambda_to_lambda_old<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, nx, ny, nz);
        hipDeviceSynchronize();
        // SOR part
        int offset = 0;   // red nodes
        // Invoke red-black SOR kernel for red nodes
        SOR_RB<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda, nx, ny, nz, omega, A, B, dx, d_e, d_f, d_g, d_h, d_m, d_n, d_R, offset);
        hipDeviceSynchronize();
        offset = 1;    // black nodes
        // Invoke red-black SOR kernel for black nodes
        SOR_RB<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda, nx, ny, nz, omega, A, B, dx, d_e, d_f, d_g, d_h, d_m, d_n, d_R,offset);
        hipDeviceSynchronize();
        dim3 numberOfBlocks2(ceil(((nx-1)*(ny-1))/(double) (BLOCKSIZE)),1,1);
        // Invoke kernel to apply Neumann boundary condition (lambda (@k=0) = lambda (@k=1))
        applyNeumannBC<<<numberOfBlocks2,numberOfThreadsPerBlock>>>(d_lambda, nx, ny);
        hipDeviceSynchronize();
        // Error calculation
        calculateError<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda,d_lambda_old, nx, ny, nz, d_value,d_bvalue);
        hipDeviceSynchronize();

        iter += 1;
        
    }
    printf("number of iteration = %d\n", iter);
    printf("error = %2.9f\n", error);
    dim3 numberOfBlocks3(ceil((nx*ny*nz)/(double) (BLOCKSIZE)),1,1);
    // Invoke final velocity (Euler) kernel
<<<<<<< HEAD
    finalVelocity<<<numberOfBlocks3,numberOfThreadsPerBlock>>>(d_u0,d_v0,d_w0,d_lambda,d_u,d_v,d_w,d_icellflag,d_f,d_h,d_n,alpha1,alpha2,dx,dy,dz,nx,ny,nz);
=======
    finalVelocity<<<numberOfBlocks3,numberOfThreadsPerBlock>>>(d_u0,d_v0,d_w0,d_lambda,d_u,d_v,d_w,d_icellflag,alpha1,alpha2,dx,dy,dz,nx,ny,nz);
>>>>>>> origin/doxygenAdd
}



void DynamicParallelism::solve(NetCDFData* netcdfDat, bool solveWind) 
{
    
<<<<<<< HEAD
	int numblocks = (numcell_cent/BLOCKSIZE)+1;
    double *value, *bvalue;
    value = new double [numcell_cent];
    bvalue = new double [numblocks];   
	double *d_u0, *d_v0, *d_w0; 
	double *d_value,*d_bvalue;
    float *d_x,*d_y,*d_z;
    double *d_u, *d_v, *d_w;  
	int *d_icellflag;
/*    nx += 1;        /// +1 for Staggered grid
=======
  
    nx += 1;        /// +1 for Staggered grid
>>>>>>> origin/doxygenAdd
    ny += 1;        /// +1 for Staggered grid
    nz += 2;        /// +2 for staggered grid and ghost cell

    long numcell_cent = (nx-1)*(ny-1)*(nz-1);         /// Total number of cell-centered values in domain
    long numface_cent = nx*ny*nz;                     /// Total number of face-centered values in domain
    
    float *x, *y, *z;
    x = new float [nx];
    y = new float [ny];
    z = new float [nz];  

    // Declare coefficients for SOR solver
<<<<<<< HEAD
	std::vector<float> e;
	std::vector<float> f;
	std::vector<float> g;
	std::vector<float> h;
	std::vector<float> m;
	std::vector<float> n;


    /// Declaration of initial wind components (u0,v0,w0)
    std::vector<double> u0;
    std::vector<double> v0;
    std::vector<double> w0;
    std::vector<double> R;    

    /// Declaration of final velocity field components (u,v,w)
    std::vector<double> u;
    std::vector<double> v;
    std::vector<double> w;

    /// Declaration of Lagrange multipliers
    std::vector<double> lambda;
	std::vector<double> lambda_old;
=======
 float *e, *f, *g, *h, *m, *n;
    e = new float [numcell_cent];
    f = new float [numcell_cent];
    g = new float [numcell_cent];
    h = new float [numcell_cent];
    m = new float [numcell_cent];
    n = new float [numcell_cent];

    float *d_e, *d_f, *d_g, *d_h, *d_m, *d_n;
    hipMalloc((void **) &d_e, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_f, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_g, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_h, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_m, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_n, numcell_cent * sizeof(float));

    /// Declaration of initial wind components (u0,v0,w0)
    double *u0, *v0, *w0;
    u0 = new double [numface_cent];
    v0 = new double [numface_cent];
    w0 = new double [numface_cent];
    
    
    double *R, *d_R;              //!> Divergence of initial velocity field
    R = new double [numcell_cent];
    hipMalloc((void **) &d_R, numcell_cent * sizeof(double));    

    /// Declaration of final velocity field components (u,v,w)
    double *u, *v, *w;
    u = new double [numface_cent];
    v = new double [numface_cent];
    w = new double [numface_cent];

    // Declare Lagrange multipliers
    /// Declaration of Lagrange multipliers
    double *lambda, *lambda_old, *d_lambda, *d_lambda_old;
    lambda = new double [numcell_cent];
    lambda_old = new double [numcell_cent];
    hipMalloc((void **) &d_lambda, numcell_cent * sizeof(double));
    hipMalloc((void **) &d_lambda_old, numcell_cent * sizeof(double));

    int numblocks = (numcell_cent/BLOCKSIZE)+1;
    double *value, *bvalue;
    value = new double [numcell_cent];
    bvalue = new double [numblocks];    
    
    
    for ( int i = 0; i < nx-1; i++){
        x[i] = (i+0.5)*dx;         /// Location of face centers in x-dir
    }
    for ( int j = 0; j < ny-1; j++){
        y[j] = (j+0.5)*dy;         /// Location of face centers in y-dir
    }
    for ( int k = 0; k < nz-1; k++){
        z[k] = (k-0.5)*dz;         /// Location of face centers in z-dir
    }
>>>>>>> origin/doxygenAdd


    float z0 = 0.1;                 /// Surface roughness
    float z_ref = 10.0;             /// Height of the measuring sensor (m)
    float U_ref = 5.0;              /// Measured velocity at the sensor height (m/s)
    //float H = 20.0;                 /// Building height
    //float W = 20.0;                 /// Building width
    //float L = 20.0;                 /// Building length
    float x_start = 90.0;           /// Building start location in x-direction
    float y_start = 90.0;           /// Building start location in y-direction
    float i_start = std::round(x_start/dx);     /// Index of building start location in x-direction
    float i_end = std::round((x_start+20.0)/dx);   /// Index of building end location in x-direction
    float j_start = std::round(y_start/dy);     /// Index of building start location in y-direction
    float j_end = std::round((y_start+20.0)/dy);   /// Index of building end location in y-direction 
    float k_end = std::round(20.0/dz);             /// Index of building end location in z-direction
<<<<<<< HEAD
    int *d_icellflag;
    std::vector<double> icellflag;       /// Cell index flag (0 = building, 1 = fluid)
=======
    int *icellflag, *d_icellflag;
    icellflag = new int [numcell_cent];       /// Cell index flag (0 = building, 1 = fluid)
>>>>>>> origin/doxygenAdd

    std::cout << "i_start:" << i_start << "\n";   // Print the number of iterations
    std::cout << "i_end:" << i_end << "\n";       // Print the number of iterations
    std::cout << "j_start:" << j_start << "\n";   // Print the number of iterations
    std::cout << "j_end:" << j_end << "\n";       // Print the number of iterations    
    std::cout << "k_end:" << k_end << "\n";       // Print the number of iterations 

    for ( int k = 0; k < nz-1; k++){
        for (int j = 0; j < ny-1; j++){
            for (int i = 0; i < nx-1; i++){

                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);            /// Lineralized index for cell centered values
<<<<<<< HEAD
				e.push_back(1.0);
				f.push_back(1.0);	
				g.push_back(1.0);
				h.push_back(1.0);
				m.push_back(1.0);
				n.push_back(1.0);
				R.push_back(0.0);
				icellflag.push_back(1);
				lambda.push_back(0.0);
				lambda_old.push_back(0.0);
               // e[icell_cent] = f[icell_cent] = g[icell_cent] = h[icell_cent] = m[icell_cent] = n[icell_cent] = 1.0;  /// Assign initial values to the coefficients for SOR solver
                //icellflag[icell_cent] = 1;                                  /// Initialize all cells to fluid   
                //lambda[icell_cent] = lambda_old[icell_cent] = 0.0;
=======
                e[icell_cent] = f[icell_cent] = g[icell_cent] = h[icell_cent] = m[icell_cent] = n[icell_cent] = 1.0;  /// Assign initial values to the coefficients for SOR solver
                icellflag[icell_cent] = 1;                                  /// Initialize all cells to fluid   
                lambda[icell_cent] = lambda_old[icell_cent] = 0.0;
>>>>>>> origin/doxygenAdd
            }
        }    
    }   

    for ( int k = 1; k < nz; k++){
        for (int j = 0; j < ny; j++){
            for (int i = 0; i < nx; i++){
                
                int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values                                
                /// Define logarithmic wind profile
<<<<<<< HEAD
				u0.push_back( U_ref*(log((z[k]+z0)/z0)/log((z_ref+z0)/z0)));
                //u0[icell_face] = U_ref*(log((z[k]+z0)/z0)/log((z_ref+z0)/z0));
				v0.push_back(0.0);
				w0.push_back(0.0);
               // v0[icell_face] = w0 [icell_face] = 0.0;         /// Perpendicular wind direction

            }
        }
    }*/

    if (mesh)
    {
        std::cout << "Creating terrain blocks...\n";
        for (int i = 0; i < nx; i++)
        {
            for (int j = 0; j < ny; j++)
            {           //get height, then add half a cell, if the height exceeds half of a cell partially, it will round up.
                float heightToMesh = mesh->getHeight(i * dx + dx * 0.5f, j * dy + dy * 0.5f) + 0.5f * dz;
                for (int k = 0; k < (int)(heightToMesh / dz); k++)
                    buildings.push_back(new RectangularBuilding(i * dx, j * dy, k * dz, dx, dy, dz));
            }
             printProgress( (float)i / (float)nx);
        }
        std::cout << "blocks created\n";
    }

	
	std::cout << "num_sites:" << num_sites << "\n";
	std::cout << "site_blayer_flag:" << site_blayer_flag[num_sites-1] << "\n";
	inputWindProfile(dx, dy, dz, nx, ny, nz, u0.data(), v0.data(), w0.data(), num_sites, site_blayer_flag.data(), site_one_overL.data(), site_xcoord.data(), site_ycoord.data(), site_wind_dir.data(), site_z0.data(), site_z_ref.data(), site_U_ref.data(), x.data(), y.data(), z.data());

    float* zm;
    zm = new float[nz];
    int* iBuildFlag;
    iBuildFlag = new int[nx*ny*nz];
    for (int i = 0; i < buildings.size(); i++)
    {
        ((RectangularBuilding*)buildings[i])->setBoundaries(dx, dy, dz, nx, ny, nz, zm, e.data(), f.data(), g.data(), h.data(), m.data(), n.data(), icellflag.data());    /// located in RectangularBuilding.h
        //((RectangularBuilding*)buildings[i])->setCells(nx, ny, nz, icellflag, iBuildFlag, i);
=======
                u0[icell_face] = U_ref*(log((z[k]+z0)/z0)/log((z_ref+z0)/z0));
                v0[icell_face] = w0 [icell_face] = 0.0;         /// Perpendicular wind direction

            }
        }
    }

    for (int k = 0; k < k_end+1; k++){
        for (int j = j_start; j < j_end; j++){
            for (int i = i_start; i < i_end; i++){

                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
                icellflag[icell_cent] = 0;                         /// Set cell index flag to building

            }
        }
    }

    for (int j = 0; j < ny-1; j++){
        for (int i = 0; i < nx-1; i++){
            int icell_cent = i + j*(nx-1);   /// Lineralized index for cell centered values
            icellflag[icell_cent] = 0.0;
        }
>>>>>>> origin/doxygenAdd
    }

   
    for (int k = 0; k < nz; k++){
        for (int j = 0; j < ny; j++){
            for (int i = 0; i < nx; i++){
                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
                int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values 
                if (icellflag[icell_cent] == 0) {
                    u0[icell_face] = 0.0;                    /// Set velocity inside the building to zero
                    u0[icell_face+1] = 0.0;
                    v0[icell_face] = 0.0;                    /// Set velocity inside the building to zero
                    v0[icell_face+nx] = 0.0;
                    w0[icell_face] = 0.0;                    /// Set velocity inside the building to zero
                    w0[icell_face+nx*ny] = 0.0;
                }
            }
        }
    }


    auto start = std::chrono::high_resolution_clock::now(); // Start recording execution time

<<<<<<< HEAD
    hipMalloc((void **) &d_e, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_f, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_g, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_h, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_m, numcell_cent * sizeof(float));
    hipMalloc((void **) &d_n, numcell_cent * sizeof(float));
	hipMalloc((void **) &d_R, numcell_cent * sizeof(double)); 
    hipMalloc((void **) &d_lambda, numcell_cent * sizeof(double));
    hipMalloc((void **) &d_lambda_old, numcell_cent * sizeof(double));
	hipMalloc((void **) &d_icellflag, numcell_cent * sizeof(int));
    hipMalloc((void **) &d_u0,numface_cent*sizeof(double));
    hipMalloc((void **) &d_v0,numface_cent*sizeof(double));
    hipMalloc((void **) &d_w0,numface_cent*sizeof(double));
    hipMalloc((void **) &d_value,numcell_cent*sizeof(double));
    hipMalloc((void **) &d_bvalue,numblocks*sizeof(double));
    hipMalloc((void **) &d_x,nx*sizeof(float));
    hipMalloc((void **) &d_y,ny*sizeof(float));
    hipMalloc((void **) &d_z,nz*sizeof(float));
    hipMalloc((void **) &d_u,numface_cent*sizeof(double));
    hipMalloc((void **) &d_v,numface_cent*sizeof(double));
    hipMalloc((void **) &d_w,numface_cent*sizeof(double));

=======
    hipMalloc((void **) &d_icellflag, numcell_cent * sizeof(int));
    hipMemcpy(d_icellflag,icellflag,numcell_cent*sizeof(int),hipMemcpyHostToDevice);
    // Allocate GPU memory
    double *d_u0, *d_v0, *d_w0;
    hipMalloc((void **) &d_u0,numface_cent*sizeof(double));
    hipMalloc((void **) &d_v0,numface_cent*sizeof(double));
    hipMalloc((void **) &d_w0,numface_cent*sizeof(double));
    // Initialize GPU input/output
    hipMemcpy(d_u0,u0,numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_v0,v0,numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_w0,w0,numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_R,R,numcell_cent*sizeof(double),hipMemcpyHostToDevice);

    /// Boundary condition for building edges
    for (int k = 1; k < nz-2; k++){
        for (int j = 1; j < ny-2; j++){
            for (int i = 1; i < nx-2; i++){
                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
                if (icellflag[icell_cent] != 0) {
                    
                    /// Wall bellow
                    if (icellflag[icell_cent-(nx-1)*(ny-1)]==0) {
                        n[icell_cent] = 0.0; 

                    }
                    /// Wall above
                    if (icellflag[icell_cent+(nx-1)*(ny-1)]==0) {
                        m[icell_cent] = 0.0;
                    }
                    /// Wall in back
                    if (icellflag[icell_cent-1]==0){
                        f[icell_cent] = 0.0; 
                    }
                    /// Wall in front
                    if (icellflag[icell_cent+1]==0){
                        e[icell_cent] = 0.0; 
                    }
                    /// Wall on right
                    if (icellflag[icell_cent-(nx-1)]==0){
                        h[icell_cent] = 0.0;
                    }
                    /// Wall on left
                    if (icellflag[icell_cent+(nx-1)]==0){
                        g[icell_cent] = 0.0; 
                    }
                }
            }
        }
    }
>>>>>>> origin/doxygenAdd

    /// New boundary condition implementation
    for (int k = 1; k < nz-1; k++){
        for (int j = 0; j < ny-1; j++){
            for (int i = 0; i < nx-1; i++){
                int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
                e[icell_cent] = e[icell_cent]/(dx*dx);
                f[icell_cent] = f[icell_cent]/(dx*dx);
                g[icell_cent] = g[icell_cent]/(dy*dy);
                h[icell_cent] = h[icell_cent]/(dy*dy);
                m[icell_cent] = m[icell_cent]/(dz*dz);
                n[icell_cent] = n[icell_cent]/(dz*dz);
            }
        }
    }
    
<<<<<<< HEAD

    hipMemcpy(d_icellflag,icellflag.data(),numcell_cent*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_u0,u0.data(),numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_v0,v0.data(),numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_w0,w0.data(),numface_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_R,R.data(),numcell_cent*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_value , value , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_bvalue , bvalue , numblocks * sizeof(double) , hipMemcpyHostToDevice);      
    hipMemcpy(d_e , e.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_f , f.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_g , g.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_h , h.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_m , m.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_n , n.data() , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_x , x.data() , nx * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_y , y.data() , ny * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_z , z.data() , nz * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_lambda , lambda.data() , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_lambda_old , lambda_old.data() , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);    


=======
    double *d_value,*d_bvalue;
    float *d_x,*d_y,*d_z;
    hipMalloc((void **) &d_value,numcell_cent*sizeof(double));
    hipMalloc((void **) &d_bvalue,numblocks*sizeof(double));
    hipMalloc((void **) &d_x,nx*sizeof(float));
    hipMalloc((void **) &d_y,ny*sizeof(float));
    hipMalloc((void **) &d_z,nz*sizeof(float));
    hipMemcpy(d_value , value , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_bvalue , bvalue , numblocks * sizeof(double) , hipMemcpyHostToDevice);      
    hipMemcpy(d_e , e , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_f , f , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_g , g , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_h , h , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_m , m , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_n , n , numcell_cent * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_x , x , nx * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_y , y , ny * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_z , z , nz * sizeof(float) , hipMemcpyHostToDevice);
    
    double *d_u, *d_v, *d_w;
    hipMalloc((void **) &d_u,numface_cent*sizeof(double));
    hipMalloc((void **) &d_v,numface_cent*sizeof(double));
    hipMalloc((void **) &d_w,numface_cent*sizeof(double));
>>>>>>> origin/doxygenAdd

    /////////////////////////////////////////////////
    //                 SOR solver              //////
    /////////////////////////////////////////////////
    
<<<<<<< HEAD

=======
    hipMemcpy(d_lambda , lambda , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_lambda_old , lambda_old , numcell_cent * sizeof(double) , hipMemcpyHostToDevice);
>>>>>>> origin/doxygenAdd
    // Invoke the main (mother) kernel
    SOR_iteration<<<1,1>>>(d_lambda,d_lambda_old, nx, ny, nz, omega, A, B, dx, d_e, d_f, d_g, d_h, d_m, d_n, d_R,itermax,tol,d_value,d_bvalue,d_u0,d_v0,d_w0,alpha1,alpha2,dy,dz,d_u,d_v,d_w,d_icellflag);
    cudaCheck(hipGetLastError()); 
    
<<<<<<< HEAD
    hipMemcpy (lambda.data() , d_lambda , numcell_cent * sizeof(double) , hipMemcpyDeviceToHost);
    hipMemcpy(u.data(),d_u,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(v.data(),d_v,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(w.data(),d_w,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
=======
    hipMemcpy (lambda , d_lambda , numcell_cent * sizeof(double) , hipMemcpyDeviceToHost);
    hipMemcpy(u,d_u,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(v,d_v,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(w,d_w,numface_cent*sizeof(double),hipMemcpyDeviceToHost);
>>>>>>> origin/doxygenAdd

    hipFree (d_lambda);
    hipFree (d_e);
    hipFree (d_f);
    hipFree (d_g);
    hipFree (d_h);
    hipFree (d_m);
    hipFree (d_n);
    hipFree (d_R);
    hipFree (d_value);
    hipFree (d_bvalue);
    hipFree (d_u0);
    hipFree (d_v0);
    hipFree (d_w0);
    hipFree (d_u);
    hipFree (d_v);
    hipFree (d_w);
    hipFree (d_x);
    hipFree (d_y);
    hipFree (d_z);
    hipFree (d_icellflag);

    auto finish = std::chrono::high_resolution_clock::now();  // Finish recording execution time
    std::chrono::duration<double> elapsed = finish - start;
    std::cout << "Elapsed time: " << elapsed.count() << " s\n";   // Print out elapsed execution time    
    
    /// Declare cell center positions
    float *x_out, *y_out, *z_out;
    x_out = new float [nx-1];
    y_out = new float [ny-1];
    z_out = new float [nz-1];


    for ( int i = 0; i < nx-1; i++) {
        x_out[i] = (i+0.5)*dx;         /// Location of cell centers in x-dir
    }
    for ( int j = 0; j < ny-1; j++){
        y_out[j] = (j+0.5)*dy;         /// Location of cell centers in y-dir
    }
    for ( int k = 0; k < nz-1; k++){
        z_out[k] = (k-0.5)*dz;         /// Location of cell centers in z-dir
    }

    /// Declare output velocity field arrays
    double ***u_out, ***v_out, ***w_out;
    u_out = new double** [nx-1];
    v_out = new double** [nx-1];
    w_out = new double** [nx-1];
    
    for (int i = 0; i < nx-1; i++){
        u_out[i] = new double* [ny-1];
        v_out[i] = new double* [ny-1];
        w_out[i] = new double* [ny-1];
        for (int j = 0; j < ny-1; j++){
            u_out[i][j] = new double [nz-1];
            v_out[i][j] = new double [nz-1];
            w_out[i][j] = new double [nz-1];
        }
    }


    for (int k = 0; k < nz-1; k++){
        for (int j = 0; j < ny-1; j++){
            for (int i = 0; i < nx-1; i++){
                int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values 
                u_out[i][j][k] = 0.5*(u[icell_face+1]+u[icell_face]);
                v_out[i][j][k] = 0.5*(v[icell_face+nx]+v[icell_face]);
                w_out[i][j][k] = 0.5*(w[icell_face+nx*ny]+w[icell_face]);
            }
        }   
    }

    // Write data to file
    ofstream outdata1;
    outdata1.open("Final velocity.dat");
    if( !outdata1 ) {                 // File couldn't be opened
        cerr << "Error: file could not be opened" << endl;
        exit(1);
    }
    // Write data to file
    for (int k = 0; k < nz; k++){
        for (int j = 0; j < ny; j++){
            for (int i = 0; i < nx; i++){
                //int icell_cent = i + j*(nx-1) + k*(nx-1)*(ny-1);   /// Lineralized index for cell centered values
                int icell_face = i + j*nx + k*nx*ny;   /// Lineralized index for cell faced values
                outdata1 << "\t" << i << "\t" << j << "\t" << k << "\t \t"<< x[i] << "\t \t" << y[j] << "\t \t" << z[k] << "\t \t"<< "\t \t" << u[icell_face] <<"\t \t"<< "\t \t"<<v[icell_face]<<"\t \t"<< "\t \t"<<w[icell_face]<< endl;   
}
        }
    }
    outdata1.close();

<<<<<<< HEAD
    netcdfDat->getData(x.data(),y.data(),z.data(),u,v,w,nx,ny,nz);
=======
    netcdfDat->getData(x,y,z,u,v,w,nx,ny,nz);
>>>>>>> origin/doxygenAdd

    // Write data to file
    ofstream outdata;
    outdata.open("Final velocity, cell-centered.dat");
    if( !outdata ) {                 // File couldn't be opened
        cerr << "Error: file could not be opened" << endl;
        exit(1);
    }
    // Write data to file
    for (int k = 0; k < nz-1; k++){
        for (int j = 0; j < ny-1; j++){
            for (int i = 0; i < nx-1; i++){
                outdata << "\t" << i << "\t" << j << "\t" << k << "\t"<< x_out[i] << "\t" << y_out[j] << "\t" << z_out[k] << "\t" << u_out[i][j][k] << "\t" << v_out[i][j][k] << "\t" << w_out[i][j][k]                     << endl;   
            }
        }
    }
    outdata.close();
}
