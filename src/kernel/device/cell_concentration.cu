#include "hip/hip_runtime.h"
/*
 * concentration.cu
 * This file is part of GPUPLUME
 *
 * Copyright (C) 2012 - Alex
 *
 * GPUPLUME is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * GPUPLUME is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with GPUPLUME. If not, see <http://www.gnu.org/licenses/>.
 */

 #ifndef __CONCENTRATION_CU_H__
 #define __CONCENTRATION_CU_H__
 
__global__ void concentration_kernel
                (float4* posPtr, uint* device_cons, const uint numParticles
// 		 , float4* debug
		) 
{ 
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  uint offset = x + y * blockDim.x * gridDim.x; 
  
  if(offset > numParticles-1) 
    return;
  
  
  float3 posf3 = make_float3(posPtr[offset]);//make_float3(43.5,55,0.66);// 
  
  int idx=(int)((posf3.x-33)/2);
  int idy=(int)((posf3.y-0)/2);
  int idz=(int)((posf3.z-0)/1.2f);
    if(posf3.x<33)
      idx=-1;
    if(posf3.y<0)
      idy=-1;
    if(posf3.y<0)
      idz=-1;
  
  if(idx>=0 && idx<60 && idy>=0 && idy<55 && idz>=0 && idz<25 )
  {
    int id=idz*55*60+idy*60+idx;
    atomicAdd(&device_cons[id], 1);
  }
  
  
//   float3 posf3 = make_float3(posPtr[offset]);//make_float3(43.5,55,0.66);// 
//   int cellIndex = (int)(posf3.z)*g_params.domain.x*g_params.domain.y + 
// 		   (int)(posf3.y)*g_params.domain.x + (int)posf3.x - 1;
// //   
//   if(cellIndex > 0 && cellIndex < g_params.domain.x*g_params.domain.y*g_params.domain.z)
//     atomicAdd(&device_cons[cellIndex], 1);
// //   else
// //     debug[offset] = make_float4(posf3, cellIndex);
}

 #endif /* __CONCENTRATION_CU_H__ */
 
