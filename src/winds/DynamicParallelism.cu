#include "hip/hip_runtime.h"
/** @file DynamicParallelism.cu */

#include "DynamicParallelism.h"

using namespace std::chrono;
using namespace std;
using std::ofstream;
using std::ifstream;
using std::istringstream;
using std::string;
using std::cerr;
using std::endl;
using std::vector;
using std::cout;
using std::to_string;

#define BLOCKSIZE 1024
#define cudaCheck(x) _cudaCheck(x, #x, __FILE__, __LINE__)

__device__ float error;


template<typename T>
void DynamicParallelism::_cudaCheck(T e, const char *func, const char *call, const int line)
{
  if (e != hipSuccess) {
    printf("\"%s\" at %d in %s\n\treturned %d\n-> %s\n", func, line, call, (int)e, hipGetErrorString(e));
    exit(EXIT_FAILURE);
  }
}

/// Divergence CUDA Kernel.
/// The divergence kernel ...
///
__global__ void divergence(float *d_u0, float *d_v0, float *d_w0, float *d_R, float *d_e, float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, int alpha1, int nx, int ny, int nz, float dx, float dy, float *d_dz_array)
{

  int icell_cent = blockDim.x * blockIdx.x + threadIdx.x;
  int k = icell_cent / ((nx - 1) * (ny - 1));
  int j = (icell_cent - k * (nx - 1) * (ny - 1)) / (nx - 1);
  int i = icell_cent - k * (nx - 1) * (ny - 1) - j * (nx - 1);
  int icell_face = i + j * nx + k * nx * ny;

  // Would be nice to figure out how to not have this branch check...
  if ((i < nx - 1) && (j < ny - 1) && (k < nz - 1) && (i >= 0) && (j >= 0) && (k > 0)) {

    // Divergence equation
    d_R[icell_cent] = (-2 * pow(alpha1, 2.0)) * (((d_e[icell_cent] * d_u0[icell_face + 1] - d_f[icell_cent] * d_u0[icell_face]) * dx) + ((d_g[icell_cent] * d_v0[icell_face + nx] - d_h[icell_cent] * d_v0[icell_face]) * dy) + (d_m[icell_cent] * d_dz_array[k] * 0.5 * (d_dz_array[k] + d_dz_array[k + 1]) * d_w0[icell_face + nx * ny] - d_n[icell_cent] * d_w0[icell_face] * d_dz_array[k] * 0.5 * (d_dz_array[k] + d_dz_array[k - 1])));
  }
}


/// Jacobi iteration.
///
///
/*__global__ void Jacobi (float *d_lambda, float *d_lambda_old, int nx, int ny, int nz, float omega, float  A, float  B, float  dx, float *d_e,
                                                float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, float *d_R)
{
    int icell_cent = blockDim.x*blockIdx.x+threadIdx.x;
    int k = icell_cent/((nx-1)*(ny-1));
    int j = (icell_cent - k*(nx-1)*(ny-1))/(nx-1);
    int i = icell_cent - k*(nx-1)*(ny-1) - j*(nx-1);

    if ( (i > 0) && (i < nx-2) && (j > 0) && (j < ny-2) && (k < nz-2) && (k > 0) ){

        d_lambda[icell_cent] = (1.0 / ( d_e[icell_cent] + d_f[icell_cent] + d_g[icell_cent] +
                                          d_h[icell_cent] + d_m[icell_cent] + d_n[icell_cent])) *
            ( d_e[icell_cent] * d_lambda_old[icell_cent+1]               + d_f[icell_cent] * d_lambda_old[icell_cent-1] +
              d_g[icell_cent] * d_lambda_old[icell_cent + (nx-1)]        + d_h[icell_cent] * d_lambda_old[icell_cent - (nx-1)] +
              d_m[icell_cent] * d_lambda_old[icell_cent + (nx-1)*(ny-1)] +
              d_n[icell_cent] * d_lambda_old[icell_cent - (nx-1)*(ny-1)] - d_R[icell_cent] );    /// Jacobi formulation
    }
}*/


/// SOR RedBlack Kernel.
///
///
__global__ void SOR_RB(float *d_lambda, float *d_lambda_old, int nx, int ny, int nz, float omega, float A, float B, float dx, float *d_e, float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, float *d_R, int offset)
{
  int icell_cent = blockDim.x * blockIdx.x + threadIdx.x;
  int k = icell_cent / ((nx - 1) * (ny - 1));
  int j = (icell_cent - k * (nx - 1) * (ny - 1)) / (nx - 1);
  int i = icell_cent - k * (nx - 1) * (ny - 1) - j * (nx - 1);

  if ((i > 0) && (i < nx - 2) && (j > 0) && (j < ny - 2) && (k < nz - 2) && (k > 0) && ((i + j + k) % 2) == offset) {

    d_lambda[icell_cent] = (omega / (d_e[icell_cent] + d_f[icell_cent] + d_g[icell_cent] + d_h[icell_cent] + d_m[icell_cent] + d_n[icell_cent])) * (d_e[icell_cent] * d_lambda[icell_cent + 1] + d_f[icell_cent] * d_lambda[icell_cent - 1] + d_g[icell_cent] * d_lambda[icell_cent + (nx - 1)] + d_h[icell_cent] * d_lambda[icell_cent - (nx - 1)] + d_m[icell_cent] * d_lambda[icell_cent + (nx - 1) * (ny - 1)] + d_n[icell_cent] * d_lambda[icell_cent - (nx - 1) * (ny - 1)] - d_R[icell_cent]) + (1.0 - omega) * d_lambda[icell_cent];/// SOR formulation
  }
}

__global__ void saveLambda(float *d_lambda, float *d_lambda_old, int nx, int ny, int nz)
{
  int ii = blockDim.x * blockIdx.x + threadIdx.x;

  if (ii < (nz - 1) * (ny - 1) * (nx - 1)) {
    d_lambda_old[ii] = d_lambda[ii];
  }
}

__global__ void applyNeumannBC(float *d_lambda, int nx, int ny)
{
  // Neumann boundary condition (lambda (@k=0) = lambda (@k=1))
  int ii = blockDim.x * blockIdx.x + threadIdx.x;

  if (ii < nx * ny) {
    d_lambda[ii] = d_lambda[ii + 1 * (nx - 1) * (ny - 1)];
  }
}

__global__ void calculateError(float *d_lambda, float *d_lambda_old, int nx, int ny, int nz, float *d_value, float *d_bvalue)
{
  int d_size = (nx - 1) * (ny - 1) * (nz - 1);
  int ii = blockDim.x * blockIdx.x + threadIdx.x;
  int numblocks = (d_size / BLOCKSIZE) + 1;

  if (ii < d_size) {
    d_value[ii] = fabs(d_lambda[ii] - d_lambda_old[ii]);
  }

  __syncthreads();

  if (threadIdx.x > 0) {
    return;
  }
  if (threadIdx.x == 0) {
    d_bvalue[blockIdx.x] = 0.0;
    for (int j = 0; j < BLOCKSIZE; j++) {
      int index = blockIdx.x * blockDim.x + j;
      if (index < d_size) {

        if (d_value[index] > d_bvalue[blockIdx.x]) {
          d_bvalue[blockIdx.x] = d_value[index];
        }
      }
    }
  }


  __syncthreads();


  if (ii > 0) {
    return;
  }

  error = 0.0;

  if (ii == 0) {
    for (int k = 0; k < numblocks; k++) {
      if (d_bvalue[k] > error) {
        error = d_bvalue[k];
      }
    }
  }
}

// Euler Final Velocity kernel
__global__ void finalVelocity(float *d_lambda, float *d_u, float *d_v, float *d_w, int *d_icellflag, float *d_f, float *d_h, float *d_n, int alpha1, int alpha2, float dx, float dy, float dz, float *d_dz_array, int nx, int ny, int nz)
{

  int icell_face = blockDim.x * blockIdx.x + threadIdx.x;
  int k = icell_face / (nx * ny);
  int j = (icell_face - k * nx * ny) / nx;
  int i = icell_face - k * nx * ny - j * nx;
  int icell_cent = i + j * (nx - 1) + k * (nx - 1) * (ny - 1);/// Lineralized index for cell centered values

  /*if ((i >= 0) && (j >= 0) && (k >= 0) && (i < nx) && (j < ny) && (k < nz - 1)) {

    d_u[icell_face] = d_u0[icell_face];
    d_v[icell_face] = d_v0[icell_face];
    d_w[icell_face] = d_w0[icell_face];
  }*/


  if ((i > 0) && (i < nx - 1) && (j > 0) && (j < ny - 1) && (k < nz - 2) && (k > 0)) {

    d_u[icell_face] = d_u[icell_face] + (1 / (2 * pow(alpha1, 2.0))) * d_f[icell_cent] * dx * (d_lambda[icell_cent] - d_lambda[icell_cent - 1]);
    d_v[icell_face] = d_v[icell_face] + (1 / (2 * pow(alpha1, 2.0))) * d_h[icell_cent] * dy * (d_lambda[icell_cent] - d_lambda[icell_cent - (nx - 1)]);
    d_w[icell_face] = d_w[icell_face] + (1 / (2 * pow(alpha2, 2.0))) * d_n[icell_cent] * d_dz_array[k] * (d_lambda[icell_cent] - d_lambda[icell_cent - (nx - 1) * (ny - 1)]);
  }


  if ((i >= 0) && (i < nx - 1) && (j >= 0) && (j < ny - 1) && (k < nz - 1) && (k >= 1) && ((d_icellflag[icell_cent] == 0) || (d_icellflag[icell_cent] == 2))) {
    d_u[icell_face] = 0;
    d_u[icell_face + 1] = 0;
    d_v[icell_face] = 0;
    d_v[icell_face + nx] = 0;
    d_w[icell_face] = 0;
    d_w[icell_face + nx * ny] = 0;
  }
}


/// SOR iteration kernel
///
__global__ void SOR_iteration(float *d_lambda, float *d_lambda_old, int nx, int ny, int nz, float omega, float A, float B, float dx, float dy, float dz, float *d_dz_array, float *d_e, float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, float *d_R, int itermax, float tol, float *d_value, float *d_bvalue, int alpha1, int alpha2, float *d_u, float *d_v, float *d_w, int *d_icellflag)
{
  int iter = 0;
  error = 1.0;

  // Calculate divergence of initial velocity field
  dim3 numberOfThreadsPerBlock(BLOCKSIZE, 1, 1);
  dim3 numberOfBlocks(ceil(((nx - 1) * (ny - 1) * (nz - 1)) / (float)(BLOCKSIZE)), 1, 1);

  // Invoke divergence kernel
  if (itermax > 0) {
    divergence<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_u, d_v, d_w, d_R, d_e, d_f, d_g, d_h, d_m, d_n, alpha1, nx, ny, nz, dx, dy, d_dz_array);
  }

  // Iterate untill convergence is reached
  while ((iter < itermax) && (error > tol) && (itermax > 0)) {

    // Save previous iteration values for error calculation
    saveLambda<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, nx, ny, nz);
    hipDeviceSynchronize();

    // SOR part
    int offset = 0;// red nodes
    // offset = ( (iter % 2) + offset ) % 2;
    //  Invoke red-black SOR kernel for red nodes
    SOR_RB<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, nx, ny, nz, omega, A, B, dx, d_e, d_f, d_g, d_h, d_m, d_n, d_R, offset);
    hipDeviceSynchronize();
    offset = 1;// black nodes
    // offset = ( (iter % 2) + offset ) % 2;
    //  Invoke red-black SOR kernel for black nodes
    SOR_RB<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, nx, ny, nz, omega, A, B, dx, d_e, d_f, d_g, d_h, d_m, d_n, d_R, offset);
    hipDeviceSynchronize();

    dim3 numberOfBlocks2(ceil(((nx - 1) * (ny - 1)) / (float)(BLOCKSIZE)), 1, 1);
    // Invoke kernel to apply Neumann boundary condition (lambda (@k=0) = lambda (@k=1))
    applyNeumannBC<<<numberOfBlocks2, numberOfThreadsPerBlock>>>(d_lambda, nx, ny);
    hipDeviceSynchronize();
    // Error calculation
    calculateError<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, nx, ny, nz, d_value, d_bvalue);
    hipDeviceSynchronize();

    iter += 1;
  }
  printf("Error = %2.9f\n", error);
  printf("Number of iteration = %d\n", iter);

  /*int i = 0;
  while (i < 300)
  {
    // Save previous iteration values for error calculation
    assign_lambda_to_lambda_old<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, nx, ny, nz);
    hipDeviceSynchronize();
    Jacobi<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, nx, ny, nz, omega, A, B, dx, d_e, d_f, d_g, d_h, d_m,
                          d_n, d_R);
    hipDeviceSynchronize();
    i += 1;
  }*/
  dim3 numberOfBlocks3(ceil((nx * ny * nz) / (float)(BLOCKSIZE)), 1, 1);
  // Invoke final velocity (Euler) kernel
  if (itermax > 0) {
    finalVelocity<<<numberOfBlocks3, numberOfThreadsPerBlock>>>(d_lambda, d_u, d_v, d_w, d_icellflag, d_f, d_h, d_n, alpha1, alpha2, dx, dy, dz, d_dz_array, nx, ny, nz);
  }
}


DynamicParallelism::DynamicParallelism(const WINDSInputData *WID, WINDSGeneralData *WGD)
  : Solver(WID, WGD)
{
  std::cout << "-------------------------------------------------------------------" << std::endl;
  std::cout << "DynamicParallelism Solver Initialization" << std::endl;
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    std::cerr << "ERROR!   hipGetDeviceCount returned "
              << static_cast<int>(error_id) << "\n\t-> "
              << hipGetErrorString(error_id) << std::endl;
    exit(EXIT_FAILURE);
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0) {
    std::cerr << "There are no available device(s) that support CUDA\n";
    exit(EXIT_FAILURE);
  } else {
    std::cout << "\tDetected " << deviceCount << " CUDA Capable device(s)" << std::endl;
  }

  int dev, driverVersion = 0, runtimeVersion = 0;

  for (dev = 0; dev < deviceCount; ++dev) {

    hipSetDevice(dev);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    std::cout << "\tDevice " << dev << ": " << deviceProp.name << std::endl;

    // Console log
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    std::cout << "\t\tCUDA Driver Version / Runtime Version: "
              << driverVersion / 1000 << "." << (driverVersion % 100) / 10 << " / "
              << runtimeVersion / 1000 << "." << (runtimeVersion % 100) / 10 << std::endl;

    std::cout << "\t\tCUDA Capability Major/Minor version number: "
              << deviceProp.major << "." << deviceProp.minor << std::endl;

    char msg[256];
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    sprintf_s(msg, sizeof(msg),
              "\t\tTotal amount of global memory: %.0f MBytes "
              "(%llu bytes)\n",
              static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
              (unsigned long long)deviceProp.totalGlobalMem);
#else
    snprintf(msg, sizeof(msg),
             "\t\tTotal amount of global memory: %.0f MBytes "
             "(%llu bytes)\n",
             static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
             (unsigned long long)deviceProp.totalGlobalMem);
#endif
    std::cout << msg;

    //    printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
    //           deviceProp.multiProcessorCount,
    //           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
    //           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
    //           deviceProp.multiProcessorCount);

    std::cout << "\t\tGPU Max Clock rate:  "
              << deviceProp.clockRate * 1e-3f << " MHz ("
              << deviceProp.clockRate * 1e-6f << " GHz)" << std::endl;

    std::cout << "\t\tPCI: BusID=" << deviceProp.pciBusID << ", "
              << "DeviceID=" << deviceProp.pciDeviceID << ", "
              << "DomainID=" << deviceProp.pciDomainID << std::endl;
  }
  hipSetDevice(0);

  char msg[256];
  int numblocks = (WGD->numcell_cent / BLOCKSIZE) + 1;
  long long memory_req = (10 * WGD->numcell_cent + 6 * WGD->numcell_face + numblocks + (WGD->nz - 1)) * sizeof(float)
                         + (WGD->numcell_cent) * sizeof(int) + 2308964352;

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
  sprintf_s(msg, sizeof(msg),
            "Total global memory required for running this case: %.0f MBytes "
            "(%llu bytes)\n",
            static_cast<float>(memory_req / 1048576.0f),
            (unsigned long long)memory_req);
#else
  snprintf(msg, sizeof(msg),
           "Total global memory required for running this case: %.0f MBytes "
           "(%llu bytes)\n",
           static_cast<float>(memory_req / 1048576.0f),
           (unsigned long long)memory_req);
#endif
  std::cout << msg;


  std::cout << "-------------------------------------------------------------------" << std::endl;
}


void DynamicParallelism::solve(const WINDSInputData *WID, WINDSGeneralData *WGD, bool solveWind)
{
  auto startTotal = std::chrono::high_resolution_clock::now();// Start
  // recording
  // execution
  // time
  itermax = WID->simParams->maxIterations;
  int numblocks = (WGD->numcell_cent / BLOCKSIZE) + 1;

  std::vector<float> value(WGD->numcell_cent, 0.0);
  std::vector<float> bvalue(numblocks, 0.0);
  float *d_u0, *d_v0, *d_w0;
  float *d_value, *d_bvalue;
  // float *d_x,*d_y,*d_z;
  float *d_u, *d_v, *d_w;
  int *d_icellflag;
  float *d_dz_array;

  auto start = std::chrono::high_resolution_clock::now();// Start recording execution time

  hipMalloc((void **)&d_e, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_f, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_g, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_h, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_m, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_n, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_R, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_lambda, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_lambda_old, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_icellflag, WGD->numcell_cent * sizeof(int));
  /*hipMalloc((void **)&d_u0, WGD->numcell_face * sizeof(float));
  hipMalloc((void **)&d_v0, WGD->numcell_face * sizeof(float));
  hipMalloc((void **)&d_w0, WGD->numcell_face * sizeof(float));*/
  hipMalloc((void **)&d_value, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_bvalue, numblocks * sizeof(float));
  hipMalloc((void **)&d_dz_array, (WGD->nz - 1) * sizeof(float));
  hipMalloc((void **)&d_u, WGD->numcell_face * sizeof(float));
  hipMalloc((void **)&d_v, WGD->numcell_face * sizeof(float));
  hipMalloc((void **)&d_w, WGD->numcell_face * sizeof(float));

#if 0
  long long memory_req = (10 * WGD->numcell_cent + 6 * WGD->numcell_face + numblocks + (WGD->nz - 1)) * sizeof(float) + (WGD->numcell_cent) * sizeof(int) + 2308964352;
  char msg[256];
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
  sprintf_s(msg, sizeof(msg),
            "  Total global memory required for running this case:                 %.0f MBytes "
            "(%llu bytes)\n",
            static_cast<float>(memory_req / 1048576.0f),
            (unsigned long long)memory_req);
#else
  snprintf(msg, sizeof(msg),
           "  Total global memory required for running this case:                 %.0f MBytes "
           "(%llu bytes)\n",
           static_cast<float>(memory_req / 1048576.0f),
           (unsigned long long)memory_req);
#endif
  std::cout << msg;
#endif

  hipMemcpy(d_icellflag, WGD->icellflag.data(), WGD->numcell_cent * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_u, WGD->u0.data(), WGD->numcell_face * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v, WGD->v0.data(), WGD->numcell_face * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w, WGD->w0.data(), WGD->numcell_face * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_R, R.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_value, value.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_bvalue, bvalue.data(), numblocks * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_e, WGD->e.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_f, WGD->f.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_g, WGD->g.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_h, WGD->h.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_m, WGD->m.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_n, WGD->n.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_dz_array, WGD->dz_array.data(), (WGD->nz - 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_lambda, lambda.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_lambda_old, lambda_old.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);

  /////////////////////////////////////////////////
  //                 SOR solver              //////
  /////////////////////////////////////////////////

  // Invoke the main (mother) kernel
  SOR_iteration<<<1, 1>>>(d_lambda, d_lambda_old, WGD->nx, WGD->ny, WGD->nz, omega, A, B, WGD->dx, WGD->dy, WGD->dz, d_dz_array, d_e, d_f, d_g, d_h, d_m, d_n, d_R, itermax, tol, d_value, d_bvalue, alpha1, alpha2, d_u, d_v, d_w, d_icellflag);
  cudaCheck(hipGetLastError());

  // hipMemcpy (lambda.data() , d_lambda , WGD->numcell_cent * sizeof(float) , hipMemcpyDeviceToHost);
  hipMemcpy(WGD->u.data(), d_u, WGD->numcell_face * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(WGD->v.data(), d_v, WGD->numcell_face * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(WGD->w.data(), d_w, WGD->numcell_face * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(lambda.data(), d_lambda, WGD->numcell_cent * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(lambda_old.data(), d_lambda_old, WGD->numcell_cent * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(R.data(), d_R, WGD->numcell_cent * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_lambda);
  hipFree(d_lambda_old);
  hipFree(d_e);
  hipFree(d_f);
  hipFree(d_g);
  hipFree(d_h);
  hipFree(d_m);
  hipFree(d_n);
  hipFree(d_R);
  hipFree(d_value);
  hipFree(d_bvalue);
  /*hipFree(d_u0);
  hipFree(d_v0);
  hipFree(d_w0);*/
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
  hipFree(d_dz_array);
  hipFree(d_icellflag);

  auto finish = std::chrono::high_resolution_clock::now();// Finish recording execution time

  std::chrono::duration<float> elapsed = finish - start;
  std::cout << "Elapsed time: " << elapsed.count() << " s\n";// Print out elapsed execution time

  /*for (int iter = 0; iter < 20; iter++ )
  {
    for (int k = 1; k < WGD->nz-2; k++)
    {
      for (int j = 1; j < WGD->ny-2; j++)
      {
          for (int i = 1; i < WGD->nx-2; i++)
          {

              int icell_cent = i + j*(WGD->nx-1) + k*(WGD->nx-1)*(WGD->ny-1);   /// Lineralized index for cell centered values

              lambda[icell_cent] = (omega / ( WGD->e[icell_cent] + WGD->f[icell_cent] + WGD->g[icell_cent] +
                                                WGD->h[icell_cent] + WGD->m[icell_cent] + WGD->n[icell_cent])) *
                    ( WGD->e[icell_cent] * lambda[icell_cent+1]        + WGD->f[icell_cent] * lambda[icell_cent-1] +
                      WGD->g[icell_cent] * lambda[icell_cent + (WGD->nx-1)] + WGD->h[icell_cent] * lambda[icell_cent-(WGD->nx-1)] +
                      WGD->m[icell_cent] * lambda[icell_cent+(WGD->nx-1)*(WGD->ny-1)] +
                      WGD->n[icell_cent] * lambda[icell_cent-(WGD->nx-1)*(WGD->ny-1)] - R[icell_cent] ) +
                    (1.0 - omega) * lambda[icell_cent];    /// SOR formulation

            }
        }
    }
  }

  for (int k = 0; k < WGD->nz-1; k++)
  {
      for (int j = 0; j < WGD->ny; j++)
      {
          for (int i = 0; i < WGD->nx; i++)
          {
              int icell_face = i + j*WGD->nx + k*WGD->nx*WGD->ny;   /// Lineralized index for cell faced values
              WGD->u[icell_face] = WGD->u0[icell_face];
              WGD->v[icell_face] = WGD->v0[icell_face];
              WGD->w[icell_face] = WGD->w0[icell_face];
          }
      }
  }


  // /////////////////////////////////////////////
  /// Update velocity field using Euler equations
  // /////////////////////////////////////////////
  for (int k = 1; k < WGD->nz-2; k++)
  {
      for (int j = 1; j < WGD->ny-1; j++)
      {
          for (int i = 1; i < WGD->nx-1; i++)
          {
              int icell_cent = i + j*(WGD->nx-1) + k*(WGD->nx-1)*(WGD->ny-1);   /// Lineralized index for cell centered values
              int icell_face = i + j*WGD->nx + k*WGD->nx*WGD->ny;               /// Lineralized index for cell faced values

              WGD->u[icell_face] = WGD->u0[icell_face] + (1/(2*pow(alpha1, 2.0))) *
                  WGD->f[icell_cent]*WGD->dx*(lambda[icell_cent]-lambda[icell_cent-1]);

                  // Calculate correct wind velocity
              WGD->v[icell_face] = WGD->v0[icell_face] + (1/(2*pow(alpha1, 2.0))) *
                  WGD->h[icell_cent]*WGD->dy*(lambda[icell_cent]-lambda[icell_cent - (WGD->nx-1)]);

              WGD->w[icell_face] = WGD->w0[icell_face]+(1/(2*pow(alpha2, 2.0))) *
                  WGD->n[icell_cent]*WGD->dz_array[k]*(lambda[icell_cent]-lambda[icell_cent - (WGD->nx-1)*(WGD->ny-1)]);
          }
      }
  }

  for (int k = 1; k < WGD->nz-1; k++)
  {
      for (int j = 0; j < WGD->ny-1; j++)
      {
          for (int i = 0; i < WGD->nx-1; i++)
          {
              int icell_cent = i + j*(WGD->nx-1) + k*(WGD->nx-1)*(WGD->ny-1);   /// Lineralized index for cell centered values
              int icell_face = i + j*WGD->nx + k*WGD->nx*WGD->ny;               /// Lineralized index for cell faced values

              // If we are inside a building, set velocities to 0.0
              if (WGD->icellflag[icell_cent] == 0 || WGD->icellflag[icell_cent] == 2)
              {
                  /// Setting velocity field inside the building to zero
                  WGD->u[icell_face] = 0;
                  WGD->u[icell_face+1] = 0;
                  WGD->v[icell_face] = 0;
                  WGD->v[icell_face+WGD->nx] = 0;
                  WGD->w[icell_face] = 0;
                  WGD->w[icell_face+WGD->nx*WGD->ny] = 0;
              }
          }
      }
  }*/
}
