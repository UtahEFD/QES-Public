#include "hip/hip_runtime.h"
/****************************************************************************
 * Copyright (c) 2024 University of Utah
 * Copyright (c) 2024 University of Minnesota Duluth
 *
 * Copyright (c) 2024 Behnam Bozorgmehr
 * Copyright (c) 2024 Jeremy A. Gibbs
 * Copyright (c) 2024 Fabien Margairaz
 * Copyright (c) 2024 Eric R. Pardyjak
 * Copyright (c) 2024 Zachary Patterson
 * Copyright (c) 2024 Rob Stoll
 * Copyright (c) 2024 Lucas Ulmer
 * Copyright (c) 2024 Pete Willemsen
 *
 * This file is part of QES-Winds
 *
 * GPL-3.0 License
 *
 * QES-Winds is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3 of the License.
 *
 * QES-Winds is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with QES-Winds. If not, see <https://www.gnu.org/licenses/>.
 ***************************************************************************/

/**
 * @file GlobalMemory.cu
 * @brief Child class of the Solver that runs the convergence
 * algorithm using DynamicParallelism on a single GPU.
 *
 * @sa Solver
 * @sa DynamicParallelism
 */

#include "GlobalMemory.h"

using namespace std::chrono;
using namespace std;
using std::ofstream;
using std::ifstream;
using std::istringstream;
using std::string;
using std::cerr;
using std::endl;
using std::vector;
using std::cout;
using std::to_string;

#define BLOCKSIZE 1024
#define cudaCheck(x) _cudaCheck(x, #x, __FILE__, __LINE__)


template<typename T>
void GlobalMemory::_cudaCheck(T e, const char *func, const char *call, const int line)
{
  if (e != hipSuccess) {
    printf("\"%s\" at %d in %s\n\treturned %d\n-> %s\n", func, line, call, (int)e, hipGetErrorString(e));
    exit(EXIT_FAILURE);
  }
}


__device__ __forceinline__ float atomicMax(float *address, float val)
{
  int ret = __float_as_int(*address);
  while (val > __int_as_float(ret)) {
    int old = ret;
    if ((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
      break;
  }
  return __int_as_float(ret);
}

// Divergence CUDA Kernel.
// The divergence kernel ...
//
__global__ void divergenceGlobal(float *d_u0, float *d_v0, float *d_w0, float *d_R, float *d_e, float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, int alpha1, int nx, int ny, int nz, float dx, float dy, float *d_dz_array)
{

  int id = blockDim.x * blockIdx.x + threadIdx.x;
  int k = id / ((nx - 1) * (ny - 1));
  int j = (id - k * (nx - 1) * (ny - 1)) / (nx - 1);
  int i = id - k * (nx - 1) * (ny - 1) - j * (nx - 1);
  int icell_face = i + j * nx + k * nx * ny;

  // Would be nice to figure out how to not have this branch check...
  if ((i < nx - 1) && (j < ny - 1) && (k < nz - 1) && (i >= 0) && (j >= 0) && (k > 0)) {

    // Divergence equation
    d_R[id] = (-2 * pow(alpha1, 2.0))
              * (((d_e[id] * d_u0[icell_face + 1] - d_f[id] * d_u0[icell_face]) * dx)
                 + ((d_g[id] * d_v0[icell_face + nx] - d_h[id] * d_v0[icell_face]) * dy)
                 + (d_m[id] * d_dz_array[k] * 0.5 * (d_dz_array[k] + d_dz_array[k + 1]) * d_w0[icell_face + nx * ny]
                    - d_n[id] * d_w0[icell_face] * d_dz_array[k] * 0.5 * (d_dz_array[k] + d_dz_array[k - 1])));
  }
}


// SOR RedBlack Kernel.
//
//
__global__ void SOR_RB_Global(float *d_lambda, int nx, int ny, int nz, float omega, float A, float B, float *d_e, float *d_f, float *d_g, float *d_h, float *d_m, float *d_n, float *d_R, int offset)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  int k = id / ((nx - 1) * (ny - 1));
  int j = (id - k * (nx - 1) * (ny - 1)) / (nx - 1);
  int i = id - k * (nx - 1) * (ny - 1) - j * (nx - 1);

  if ((i > 0) && (i < nx - 2) && (j > 0) && (j < ny - 2) && (k < nz - 2) && (k > 0) && ((i + j + k) % 2) == offset) {
    // SOR formulation
    d_lambda[id] = (omega / (d_e[id] + d_f[id] + d_g[id] + d_h[id] + d_m[id] + d_n[id]))
                     * (d_e[id] * d_lambda[id + 1]
                        + d_f[id] * d_lambda[id - 1]
                        + d_g[id] * d_lambda[id + (nx - 1)]
                        + d_h[id] * d_lambda[id - (nx - 1)]
                        + d_m[id] * d_lambda[id + (nx - 1) * (ny - 1)]
                        + d_n[id] * d_lambda[id - (nx - 1) * (ny - 1)]
                        - d_R[id])
                   + (1.0 - omega) * d_lambda[id];
  }
}


__global__ void saveLambdaGlobal(float *d_lambda, float *d_lambda_old, int d_size)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  if (id < d_size) {
    d_lambda_old[id] = d_lambda[id];
  }
}

__global__ void applyNeumannBCGlobal(float *d_lambda, int nx, int ny)
{
  // Neumann boundary condition (lambda (@k=0) = lambda (@k=1))
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  if (id < nx * ny) {
    d_lambda[id] = d_lambda[id + 1 * (nx - 1) * (ny - 1)];
  }
}

__global__ void calculateErrorGlobal(float *d_lambda, float *d_lambda_old, int nx, int ny, int nz, float *d_value, float *d_bvalue, float *error)
{

  int d_size = (nx - 1) * (ny - 1) * (nz - 1);
  int ii = blockDim.x * blockIdx.x + threadIdx.x;
  int numblocks = (d_size / BLOCKSIZE) + 1;

  if (ii < d_size) {
    d_value[ii] = fabs(d_lambda[ii] - d_lambda_old[ii]);
  }

  __syncthreads();

  if (threadIdx.x > 0) {
    return;
  }
  if (threadIdx.x == 0) {
    d_bvalue[blockIdx.x] = 0.0;
    for (int j = 0; j < BLOCKSIZE; j++) {
      int index = blockIdx.x * blockDim.x + j;
      if (index < d_size) {

        if (d_value[index] > d_bvalue[blockIdx.x]) {
          d_bvalue[blockIdx.x] = d_value[index];
        }
      }
    }
  }


  __syncthreads();


  if (ii > 0) {
    return;
  }

  error[0] = 0.0;

  if (ii == 0) {
    for (int k = 0; k < numblocks; k++) {
      if (d_bvalue[k] > error[0]) {
        error[0] = d_bvalue[k];
      }
    }
  }
}

__global__ void calculateErrorGlobal2(float *d_lambda, float *d_lambda_old, float *d_value, int d_size)
{
  int ii = blockDim.x * blockIdx.x + threadIdx.x;
  // int numblocks = (d_size / BLOCKSIZE) + 1;

  if (ii < d_size) {
    float error = fabs(d_lambda[ii] - d_lambda_old[ii]);

    // atomicMAX using atomicCAS
    int r = __float_as_int(*d_value);
    while (error > __int_as_float(r)) {
      int o = r;
      if ((r = atomicCAS((int *)d_value, o, __float_as_int(error))) == o)
        break;
    }
    error = __int_as_float(r);
  }
}

// Euler Final Velocity kernel
__global__ void finalVelocityGlobal(float *d_lambda, float *d_u, float *d_v, float *d_w, int *d_icellflag, float *d_f, float *d_h, float *d_n, int alpha1, int alpha2, float dx, float dy, float dz, float *d_dz_array, int nx, int ny, int nz)
{

  int icell_face = blockDim.x * blockIdx.x + threadIdx.x;
  int k = icell_face / (nx * ny);
  int j = (icell_face - k * nx * ny) / nx;
  int i = icell_face - k * nx * ny - j * nx;
  int icell_cent = i + j * (nx - 1) + k * (nx - 1) * (ny - 1);// Lineralized index for cell centered values


  if ((i > 0) && (i < nx - 1) && (j > 0) && (j < ny - 1) && (k < nz - 2) && (k > 0)) {

    d_u[icell_face] = d_u[icell_face] + (1 / (2 * pow(alpha1, 2.0))) * d_f[icell_cent] * dx * (d_lambda[icell_cent] - d_lambda[icell_cent - 1]);
    d_v[icell_face] = d_v[icell_face] + (1 / (2 * pow(alpha1, 2.0))) * d_h[icell_cent] * dy * (d_lambda[icell_cent] - d_lambda[icell_cent - (nx - 1)]);
    d_w[icell_face] = d_w[icell_face] + (1 / (2 * pow(alpha2, 2.0))) * d_n[icell_cent] * d_dz_array[k] * (d_lambda[icell_cent] - d_lambda[icell_cent - (nx - 1) * (ny - 1)]);
  }

  if ((i >= 0) && (i < nx - 1) && (j >= 0) && (j < ny - 1) && (k < nz - 1) && (k >= 1) && ((d_icellflag[icell_cent] == 0) || (d_icellflag[icell_cent] == 2))) {
    d_u[icell_face] = 0;
    d_u[icell_face + 1] = 0;
    d_v[icell_face] = 0;
    d_v[icell_face + nx] = 0;
    d_w[icell_face] = 0;
    d_w[icell_face + nx * ny] = 0;
  }
}


void GlobalMemory::solve(const WINDSInputData *WID, WINDSGeneralData *WGD, bool solveWind)
{

  itermax = WID->simParams->maxIterations;
  // int numblocks = (WGD->numcell_cent / BLOCKSIZE) + 1;
  R.resize(WGD->numcell_cent, 0.0);

  std::cout << "[Solver] Running Global Memory Solver (GPU) ..." << std::endl;

  // std::vector<float> value(WGD->numcell_cent, 0.0);
  // std::vector<float> bvalue(numblocks, 0.0);

  float *d_u, *d_v, *d_w;
  // float *d_value, *d_bvalue;
  int *d_icellflag;
  float *d_dz_array;
  float *d_error;

  auto start = std::chrono::high_resolution_clock::now();// Start recording execution time

  hipMalloc((void **)&d_dz_array, (WGD->nz - 1) * sizeof(float));

  hipMalloc((void **)&d_e, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_f, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_g, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_h, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_m, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_n, WGD->numcell_cent * sizeof(float));

  hipMalloc((void **)&d_lambda, WGD->numcell_cent * sizeof(float));
  hipMalloc((void **)&d_lambda_old, WGD->numcell_cent * sizeof(float));

  hipMalloc((void **)&d_R, WGD->numcell_cent * sizeof(float));

  // hipMalloc((void **)&d_value, WGD->numcell_cent * sizeof(float));
  // hipMalloc((void **)&d_bvalue, numblocks * sizeof(float));

  hipMalloc((void **)&d_u, WGD->numcell_face * sizeof(float));
  hipMalloc((void **)&d_v, WGD->numcell_face * sizeof(float));
  hipMalloc((void **)&d_w, WGD->numcell_face * sizeof(float));

  hipMalloc((void **)&d_icellflag, WGD->numcell_cent * sizeof(int));

  hipMemcpy(d_u, WGD->u0.data(), WGD->numcell_face * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v, WGD->v0.data(), WGD->numcell_face * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w, WGD->w0.data(), WGD->numcell_face * sizeof(float), hipMemcpyHostToDevice);

  hipMemset(d_R, 0.0, sizeof(float));
  // hipMemcpy(d_R, R.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_e, WGD->e.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_f, WGD->f.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_g, WGD->g.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_h, WGD->h.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_m, WGD->m.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_n, WGD->n.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_dz_array, WGD->dz_array.data(), (WGD->nz - 1) * sizeof(float), hipMemcpyHostToDevice);

  // hipMemcpy(d_value, value.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  // hipMemcpy(d_bvalue, bvalue.data(), numblocks * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_lambda, lambda.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_lambda_old, lambda_old.data(), WGD->numcell_cent * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_icellflag, WGD->icellflag.data(), WGD->numcell_cent * sizeof(int), hipMemcpyHostToDevice);

  dim3 numberOfThreadsPerBlock(BLOCKSIZE, 1, 1);
  dim3 numberOfBlocks(ceil(((WGD->nx - 1) * (WGD->ny - 1) * (WGD->nz - 1)) / (float)(BLOCKSIZE)), 1, 1);

  // Invoke divergence kernel
  divergenceGlobal<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_u, d_v, d_w, d_R, d_e, d_f, d_g, d_h, d_m, d_n, alpha1, WGD->nx, WGD->ny, WGD->nz, WGD->dx, WGD->dy, d_dz_array);


  /////////////////////////////////////////////////
  //                 SOR solver              //////
  /////////////////////////////////////////////////

  int iter = 0;
  // float error;
  // std::vector<float> max_error(1, 1.0);
  float max_error = 1.0;

  hipMalloc((void **)&d_error, 1 * sizeof(float));
  // hipMemcpy(d_error, max_error.data(), 1 * sizeof(float), hipMemcpyHostToDevice);


  // Main solver loop
  while ((iter < itermax) && (max_error > tol)) {
    // Save previous iteration values for error calculation
    saveLambdaGlobal<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, WGD->numcell_cent);
    cudaCheck(hipGetLastError());

    // Red nodes pass
    SOR_RB_Global<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_lambda, WGD->nx, WGD->ny, WGD->nz, omega, A, B, d_e, d_f, d_g, d_h, d_m, d_n, d_R, 0);
    cudaCheck(hipGetLastError());

    // Black nodes pass
    SOR_RB_Global<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_lambda, WGD->nx, WGD->ny, WGD->nz, omega, A, B, d_e, d_f, d_g, d_h, d_m, d_n, d_R, 1);
    cudaCheck(hipGetLastError());

    dim3 numberOfBlocks2(ceil(((WGD->nx - 1) * (WGD->ny - 1)) / (float)(BLOCKSIZE)), 1, 1);
    // Invoke kernel to apply Neumann boundary condition (lambda (@k=0) = lambda (@k=1))
    applyNeumannBCGlobal<<<numberOfBlocks2, numberOfThreadsPerBlock>>>(d_lambda, WGD->nx, WGD->ny);

    // calculateErrorGlobal<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, WGD->nx, WGD->ny, WGD->nz, d_value, d_bvalue, d_error);
    hipMemset(d_error, 0, sizeof(float));
    calculateErrorGlobal2<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_lambda, d_lambda_old, d_error, WGD->numcell_cent);
    hipMemcpy(&max_error, d_error, 1 * sizeof(float), hipMemcpyDeviceToHost);
    iter += 1;
  }

  printf("[Solver] Residual after %d itertations: %2.9f\n", iter, max_error);
  // std::cout << "Error:" << max_error[0] << "\n";
  // std::cout << "Number of iterations:" << iter << "\n";// Print the number of iterations

  dim3 numberOfBlocks3(ceil((WGD->nx * WGD->ny * WGD->nz) / (float)(BLOCKSIZE)), 1, 1);
  // Invoke final velocity (Euler) kernel
  finalVelocityGlobal<<<numberOfBlocks3, numberOfThreadsPerBlock>>>(d_lambda, d_u, d_v, d_w, d_icellflag, d_f, d_h, d_n, alpha1, alpha2, WGD->dx, WGD->dy, WGD->dz, d_dz_array, WGD->nx, WGD->ny, WGD->nz);
  cudaCheck(hipGetLastError());

  hipMemcpy(WGD->u.data(), d_u, WGD->numcell_face * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(WGD->v.data(), d_v, WGD->numcell_face * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(WGD->w.data(), d_w, WGD->numcell_face * sizeof(float), hipMemcpyDeviceToHost);


  hipFree(d_lambda);
  hipFree(d_lambda_old);
  hipFree(d_e);
  hipFree(d_f);
  hipFree(d_g);
  hipFree(d_h);
  hipFree(d_m);
  hipFree(d_n);
  hipFree(d_R);
  // hipFree(d_value);
  // hipFree(d_bvalue);
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
  hipFree(d_dz_array);
  hipFree(d_icellflag);

  auto finish = std::chrono::high_resolution_clock::now();// Finish recording execution time

  std::chrono::duration<float> elapsed = finish - start;
  std::cout << "\t\t Elapsed time: " << elapsed.count() << " s\n";// Print out elapsed execution time
}
