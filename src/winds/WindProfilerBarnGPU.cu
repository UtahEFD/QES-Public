#include "hip/hip_runtime.h"
/****************************************************************************
 * Copyright (c) 2022 University of Utah
 * Copyright (c) 2022 University of Minnesota Duluth
 *
 * Copyright (c) 2022 Behnam Bozorgmehr
 * Copyright (c) 2022 Jeremy A. Gibbs
 * Copyright (c) 2022 Fabien Margairaz
 * Copyright (c) 2022 Eric R. Pardyjak
 * Copyright (c) 2022 Zachary Patterson
 * Copyright (c) 2022 Rob Stoll
 * Copyright (c) 2022 Lucas Ulmer
 * Copyright (c) 2022 Pete Willemsen
 *
 * This file is part of QES-Winds
 *
 * GPL-3.0 License
 *
 * QES-Winds is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3 of the License.
 *
 * QES-Winds is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with QES-Winds. If not, see <https://www.gnu.org/licenses/>.
 ****************************************************************************/

/** @file Sensor.cu */

#include <cstdio>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <math.h>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "Sensor.h"

#include "WindProfilerBarnGPU.h"

#include "WINDSInputData.h"
#include "WINDSGeneralData.h"

#define BLOCKSIZE 1024
#define cudaCheck(x) _cudaCheck(x, #x, __FILE__, __LINE__)

template<typename T>
void WindProfilerBarnGPU::_cudaCheck(T e, const char *func, const char *call, const int line)
{
  if (e != hipSuccess) {
    printf("\"%s\" at %d in %s\n\treturned %d\n-> %s\n", func, line, call, (int)e, hipGetErrorString(e));
    exit(EXIT_FAILURE);
  }
}


__global__ void Calculatewm(float *d_wm, float *d_wms, float *d_sum_wm, float *d_site_xcoord, float *d_site_ycoord, float *d_x, float *d_y, float lamda, float s_gamma, int num_sites, int nx, int ny, int nz)
{

  int ii = blockDim.x * blockIdx.x + threadIdx.x;
  int j = ii / ((nx)*num_sites);
  int i = (ii - j * (nx)*num_sites) / num_sites;
  int site_id = ii - j * (nx)*num_sites - i * num_sites;

  if ((i < nx) && (j < ny) && (site_id < num_sites) && (i >= 0) && (j >= 0) && (site_id >= 0)) {
    d_wm[ii] = exp((-1 / lamda) * pow(d_site_xcoord[site_id] - d_x[i], 2.0) - (1 / lamda) * pow(d_site_ycoord[site_id] - d_y[j], 2.0));
    d_wms[ii] = exp((-1 / (s_gamma * lamda)) * pow(d_site_xcoord[site_id] - d_x[i], 2.0) - (1 / (s_gamma * lamda)) * pow(d_site_ycoord[site_id] - d_y[j], 2.0));
  }
  __syncthreads();

  if ((i < nx) && (j < ny) && (site_id < num_sites) && (i >= 0) && (j >= 0) && (site_id >= 0)) {
    if (site_id == 0) {
      for (int id = 0; id < num_sites; id++) {
        d_sum_wm[i + j * nx] = d_sum_wm[i + j * nx] + d_wm[ii + id];
      }
    }
  }

  if ((i < nx) && (j < ny) && (site_id < num_sites) && (i >= 0) && (j >= 0) && (site_id >= 0)) {
    if (site_id == 0) {
      if (d_sum_wm[i + j * nx] == 0) {
        for (int id = 0; id < num_sites; id++) {
          d_wm[ii + id] = 1e-20;
        }
      }
    }
  }
}


__global__ void CalculateInitialWind(float *d_wm, float *d_sum_wm, float *d_sum_wu, float *d_sum_wv, float *d_u0, float *d_v0, float *d_w0, float *d_u_prof, float *d_v_prof, int *d_site_id, int *d_terrain_face_id, int *d_k_mod, int num_sites, int nx, int ny, int nz, float asl_percent, int *d_abl_height, float *d_z, float *d_surf_layer_height)
{

  int ii = blockDim.x * blockIdx.x + threadIdx.x;
  int j = ii / (nx);
  int i = (ii - j * nx);

  if ((i < nx) && (j < ny) && (i >= 0) && (j >= 0)) {
    for (int k = 1; k < nz - 1; k++) {
      if (k + d_terrain_face_id[ii] < nz) {
        d_k_mod[ii] = k + d_terrain_face_id[ii] - 1;
      } else {
        continue;
      }
      int idx = i + j * nx + d_k_mod[ii] * nx * ny;
      d_sum_wu[idx] = 0;
      d_sum_wv[idx] = 0;
      d_sum_wm[i + j * nx] = 0;

      for (int id = 0; id < num_sites; id++) {
        // If the height difference between the terrain at the curent cell and sensor location is less than ABL height
        if (abs(d_z[d_terrain_face_id[ii]] - d_z[d_terrain_face_id[d_site_id[id]]]) > d_abl_height[id]) {
          d_surf_layer_height[ii] = asl_percent * d_abl_height[id];
        } else {
          d_surf_layer_height[ii] = asl_percent * (2 * d_abl_height[id] - (d_z[d_terrain_face_id[ii]] - d_z[d_terrain_face_id[d_site_id[id]]]));
        }
        // If sum of z index and the terrain index at the sensor location is outside the domain
        if (k + d_terrain_face_id[d_site_id[id]] - 1> nz - 2) {
          d_sum_wu[idx] = d_sum_wu[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * d_u_prof[nz - 2 + id * nz];
          d_sum_wv[idx] = d_sum_wv[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * d_v_prof[nz - 2 + id * nz];
          d_sum_wm[i + j * nx] = d_sum_wm[i + j * nx] + d_wm[id + i * num_sites + j * num_sites * nx];
        }// If height (above ground) is less than or equal to ASL height
        else if (d_z[k] <= d_surf_layer_height[ii]) {
          d_sum_wu[idx] = d_sum_wu[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * d_u_prof[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz];
          d_sum_wv[idx] = d_sum_wv[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * d_v_prof[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz];
          d_sum_wm[i + j * nx] = d_sum_wm[i + j * nx] + d_wm[id + i * num_sites + j * num_sites * nx];
        }// If height (above ground) is greater than ASL height and modified index is inside the domain
        else if (d_z[k] > d_surf_layer_height[ii] && k + d_terrain_face_id[d_site_id[id]] - 1 < nz && d_k_mod[ii] > k + d_terrain_face_id[d_site_id[id]] - 1) {
	  if (abs(d_z[d_terrain_face_id[ii]] - d_z[d_terrain_face_id[d_site_id[id]]]) > d_abl_height[id] || d_z[k] + d_z[d_terrain_face_id[ii]] > d_z[d_terrain_face_id[d_site_id[id]]] + d_abl_height[id] ) {
	    continue;
	  }
          d_sum_wu[idx] = d_sum_wu[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * d_u_prof[d_k_mod[ii] + id * nz];
          d_sum_wv[idx] = d_sum_wv[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * d_v_prof[d_k_mod[ii] + id * nz];
          d_sum_wm[i + j * nx] = d_sum_wm[i + j * nx] + d_wm[id + i * num_sites + j * num_sites * nx];
        } else {
	  if (abs(d_z[d_terrain_face_id[ii]] - d_z[d_terrain_face_id[d_site_id[id]]]) > d_abl_height[id] || d_z[k] + d_z[d_terrain_face_id[ii]] > d_z[d_terrain_face_id[d_site_id[id]]] + d_abl_height[id] ) {
	    continue;
	  }
          d_sum_wu[idx] = d_sum_wu[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * d_u_prof[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz];
          d_sum_wv[idx] = d_sum_wv[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * d_v_prof[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz];
          d_sum_wm[i + j * nx] = d_sum_wm[i + j * nx] + d_wm[id + i * num_sites + j * num_sites * nx];
        }
      }

      int icell_face = i + j * nx + d_k_mod[ii] * nx * ny;
      d_u0[icell_face] = d_sum_wu[idx] / d_sum_wm[i + j * nx];
      d_v0[icell_face] = d_sum_wv[idx] / d_sum_wm[i + j * nx];
      d_w0[icell_face] = 0.0;
    }
  }
}


__global__ void CalculateInit(float *d_site_xcoord, float *d_site_ycoord, float *d_x, float *d_y, float *d_dxx, float *d_dyy, int *d_iwork, int *d_jwork, float *d_u12, float *d_u34, float *d_v12, float *d_v34, float *d_u0_int, float *d_v0_int, float *d_u0, float *d_v0, float *d_u_prof, float *d_v_prof, int *d_site_id, int *d_terrain_face_id, int num_sites, float dx, float dy, int nx, int ny, int nz)
{
  int ii = blockDim.x * blockIdx.x + threadIdx.x;

  if (ii < num_sites) {
    if (d_site_xcoord[ii] > 0 && d_site_xcoord[ii] < (nx - 1) * dx && d_site_ycoord[ii] > 0 && d_site_ycoord[ii] < (ny - 1) * dy) {

      for (int j = 0; j < ny; j++) {
        if (d_y[j] < d_site_ycoord[ii]) {
          d_jwork[ii] = j;
        }
      }
      for (int i = 0; i < nx; i++) {
        if (d_x[i] < d_site_xcoord[ii]) {
          d_iwork[ii] = i;
        }
      }

      d_dxx[ii] = d_site_xcoord[ii] - d_x[d_iwork[ii]];
      d_dyy[ii] = d_site_ycoord[ii] - d_y[d_jwork[ii]];
      int index = d_iwork[ii] + d_jwork[ii] * nx;

      for (int k = d_terrain_face_id[index]; k < nz; k++) {
        int idx = k + ii * nz;
        int index_work = d_iwork[ii] + d_jwork[ii] * nx + k * nx * ny;
        d_u12[idx] = (1 - (d_dxx[ii] / dx)) * d_u0[index_work + nx] + (d_dxx[ii] / dx) * d_u0[index_work + 1 + nx];
        d_u34[idx] = (1 - (d_dxx[ii] / dx)) * d_u0[d_iwork[ii] + d_jwork[ii] * nx + k * nx * ny] + (d_dxx[ii] / dx) * d_u0[d_iwork[ii] + d_jwork[ii] * nx + k * nx * ny + 1];
        d_u0_int[idx] = (d_dyy[ii] / dy) * d_u12[idx] + (1 - (d_dyy[ii] / dy)) * d_u34[idx];


        d_v12[idx] = (1 - (d_dxx[ii] / dx)) * d_v0[index_work + nx] + (d_dxx[ii] / dx) * d_v0[index_work + 1 + nx];
        d_v34[idx] = (1 - (d_dxx[ii] / dx)) * d_v0[index_work] + (d_dxx[ii] / dx) * d_v0[index_work + 1];
        d_v0_int[idx] = (d_dyy[ii] / dy) * d_v12[idx] + (1 - (d_dyy[ii] / dy)) * d_v34[idx];
      }
    } else {
      int id;
      for (int k = 1; k < nz; k++) {
        if (k + d_terrain_face_id[d_site_id[ii]] - 1 > nz - 2) {
          id = nz - 2 + ii * nz;
          d_u0_int[id] = d_u_prof[nz - 2 + ii * nz];
          d_v0_int[id] = d_v_prof[nz - 2 + ii * nz];
        } else {
          id = k + d_terrain_face_id[d_site_id[ii]] - 1 + ii * nz;
          d_u0_int[id] = d_u_prof[k + d_terrain_face_id[d_site_id[ii]] - 1 + ii * nz];
          d_v0_int[id] = d_v_prof[k + d_terrain_face_id[d_site_id[ii]] - 1 + ii * nz];
        }
      }
    }
  }
}

__global__ void CorrectInitialWind(float *d_wm, float *d_sum_wm, float *d_sum_wu, float *d_sum_wv, float *d_u0, float *d_v0, float *d_w0, float *d_u_prof, float *d_v_prof, float *d_u0_int, float *d_v0_int, int *d_site_id, int *d_terrain_face_id, int *d_k_mod, int num_sites, int nx, int ny, int nz, float asl_percent, int *d_abl_height, float *d_z, float *d_surf_layer_height)
{

  int ii = blockDim.x * blockIdx.x + threadIdx.x;
  int j = ii / (nx);
  int i = (ii - j * nx);

  if ((i < nx) && (j < ny) && (i >= 0) && (j >= 0)) {
    for (int k = 1; k < nz - 1; k++) {
      if (k + d_terrain_face_id[ii] < nz) {
        d_k_mod[ii] = k + d_terrain_face_id[ii] - 1;
      } else {
        continue;
      }
      int idx = i + j * nx + d_k_mod[ii] * nx * ny;
      d_sum_wu[idx] = 0;
      d_sum_wv[idx] = 0;
      d_sum_wm[i + j * nx] = 0;

      for (int id = 0; id < num_sites; id++) {
        // If the height difference between the terrain at the curent cell and sensor location is less than ABL height
        if (abs(d_z[d_terrain_face_id[ii]] - d_z[d_terrain_face_id[d_site_id[id]]]) > d_abl_height[id]) {
          d_surf_layer_height[ii] = asl_percent * d_abl_height[id];
        } else {
          d_surf_layer_height[ii] = asl_percent * (2 * d_abl_height[id] - (d_z[d_terrain_face_id[ii]] - d_z[d_terrain_face_id[d_site_id[id]]]));
        }
        // If sum of z index and the terrain index at the sensor location is outside the domain
        if (k + d_terrain_face_id[d_site_id[id]] - 1 > nz - 2) {
          d_sum_wu[idx] = d_sum_wu[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * (d_u_prof[nz - 2 + id * nz] - d_u0_int[nz - 2 + id * nz]);
          d_sum_wv[idx] = d_sum_wv[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * (d_v_prof[nz - 2 + id * nz] - d_v0_int[nz - 2 + id * nz]);
          d_sum_wm[i + j * nx] = d_sum_wm[i + j * nx] + d_wm[id + i * num_sites + j * num_sites * nx];
        }// If height (above ground) is less than or equal to ASL height
        else if (d_z[k] <= d_surf_layer_height[ii]) {
          d_sum_wu[idx] = d_sum_wu[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * (d_u_prof[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz] - d_u0_int[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz]);
          d_sum_wv[idx] = d_sum_wv[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * (d_v_prof[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz] - d_v0_int[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz]);
          d_sum_wm[i + j * nx] = d_sum_wm[i + j * nx] + d_wm[id + i * num_sites + j * num_sites * nx];
        }// If height (above ground) is greater than ASL height and modified index is inside the domain
        else if (d_z[k] > d_surf_layer_height[ii] && k + d_terrain_face_id[d_site_id[id]] - 1 < nz && d_k_mod[ii] > k + d_terrain_face_id[d_site_id[id]] - 1) {
	  if (abs(d_z[d_terrain_face_id[ii]] - d_z[d_terrain_face_id[d_site_id[id]]]) > d_abl_height[id] || d_z[k] + d_z[d_terrain_face_id[ii]] > d_z[d_terrain_face_id[d_site_id[id]]] + d_abl_height[id] ) {
	    continue;
	  }
          d_sum_wu[idx] = d_sum_wu[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * (d_u_prof[d_k_mod[ii] + id * nz] - d_u0_int[d_k_mod[ii] + id * nz] );
          d_sum_wv[idx] = d_sum_wv[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * (d_v_prof[d_k_mod[ii] + id * nz] - d_v0_int[d_k_mod[ii] + id * nz]);
          d_sum_wm[i + j * nx] = d_sum_wm[i + j * nx] + d_wm[id + i * num_sites + j * num_sites * nx];
        } else {
	  if (abs(d_z[d_terrain_face_id[ii]] - d_z[d_terrain_face_id[d_site_id[id]]]) > d_abl_height[id] || d_z[k] + d_z[d_terrain_face_id[ii]] > d_z[d_terrain_face_id[d_site_id[id]]] + d_abl_height[id] ) {
	    continue;
	  }
          d_sum_wu[idx] = d_sum_wu[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * (d_u_prof[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz] - d_u0_int[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz]);
          d_sum_wv[idx] = d_sum_wv[idx] + d_wm[id + i * num_sites + j * num_sites * nx] * (d_v_prof[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz] - d_v0_int[k + d_terrain_face_id[d_site_id[id]] - 1 + id * nz]);
          d_sum_wm[i + j * nx] = d_sum_wm[i + j * nx] + d_wm[id + i * num_sites + j * num_sites * nx];
        }
      }

      if (d_sum_wm[i + j * nx] != 0) {
        int icell_face = i + j * nx + d_k_mod[ii] * nx * ny;
        d_u0[icell_face] = d_u0[icell_face] + d_sum_wu[idx] / d_sum_wm[i + j * nx];
        d_v0[icell_face] = d_v0[icell_face] + d_sum_wv[idx] / d_sum_wm[i + j * nx];
        d_w0[icell_face] = 0.0;
      }
    }
  }
}


__global__ void BarnesScheme(float *d_u_prof, float *d_v_prof, float *d_wm, float *d_wms, float *d_u0_int, float *d_v0_int, float *d_x, float *d_y, float *d_z, float *d_site_xcoord, float *d_site_ycoord, float *d_sum_wm, float *d_sum_wu, float *d_sum_wv, float *d_u0, float *d_v0, float *d_w0, int *d_iwork, int *d_jwork, int *d_site_id, int *d_terrain_face_id, int *d_k_mod, float *d_dxx, float *d_dyy, float *d_u12, float *d_u34, float *d_v12, float *d_v34, int num_sites, int nx, int ny, int nz, float dx, float dy, float asl_percent, int *d_abl_height, float *d_surf_layer_height)
{
  float rc_sum, rc_val, xc, yc, rc;
  float dn, lamda, s_gamma;
  rc_sum = 0.0;
  for (int i = 0; i < num_sites; i++) {
    rc_val = 1000000.0;
    for (int ii = 0; ii < num_sites; ii++) {
      xc = d_site_xcoord[ii] - d_site_xcoord[i];
      yc = d_site_ycoord[ii] - d_site_ycoord[i];
      rc = sqrt(pow(xc, 2.0) + pow(yc, 2.0));
      if (rc < rc_val && ii != i) {
        rc_val = rc;
      }
    }
    rc_sum = rc_sum + rc_val;
  }
  dn = rc_sum / num_sites;
  lamda = 5.052 * pow((2 * dn / M_PI), 2.0);
  s_gamma = 0.2;

  dim3 numberOfThreadsPerBlock(BLOCKSIZE, 1, 1);
  dim3 numberOfBlocks(ceil((num_sites * nx * ny) / (float)(BLOCKSIZE)), 1, 1);

  Calculatewm<<<numberOfBlocks, numberOfThreadsPerBlock>>>(d_wm, d_wms, d_sum_wm, d_site_xcoord, d_site_ycoord, d_x, d_y, lamda, s_gamma, num_sites, nx, ny, nz);
  hipDeviceSynchronize();

  dim3 numberOfThreadsPerBlock1(BLOCKSIZE, 1, 1);
  dim3 numberOfBlocks1(ceil((nx * ny) / (float)(BLOCKSIZE)), 1, 1);

  CalculateInitialWind<<<numberOfBlocks1, numberOfThreadsPerBlock1>>>(d_wm, d_sum_wm, d_sum_wu, d_sum_wv, d_u0, d_v0, d_w0, d_u_prof, d_v_prof, d_site_id, d_terrain_face_id, d_k_mod, num_sites, nx, ny, nz, asl_percent, d_abl_height, d_z, d_surf_layer_height);
  hipDeviceSynchronize();

  dim3 numberOfThreadsPerBlock2(BLOCKSIZE, 1, 1);
  dim3 numberOfBlocks2(ceil((num_sites) / (float)(BLOCKSIZE)), 1, 1);

  CalculateInit<<<numberOfBlocks2, numberOfThreadsPerBlock2>>>(d_site_xcoord, d_site_ycoord, d_x, d_y, d_dxx, d_dyy, d_iwork, d_jwork, d_u12, d_u34, d_v12, d_v34, d_u0_int, d_v0_int, d_u0, d_v0, d_u_prof, d_v_prof, d_site_id, d_terrain_face_id, num_sites, dx, dy, nx, ny, nz);
  hipDeviceSynchronize();

  CorrectInitialWind<<<numberOfBlocks1, numberOfThreadsPerBlock1>>>(d_wm, d_sum_wm, d_sum_wu, d_sum_wv, d_u0, d_v0, d_w0, d_u_prof, d_v_prof, d_u0_int, d_v0_int, d_site_id, d_terrain_face_id, d_k_mod, num_sites, nx, ny, nz, asl_percent, d_abl_height, d_z, d_surf_layer_height);
  hipDeviceSynchronize();
}

void WindProfilerBarnGPU::BarnesInterpolationGPU(const WINDSInputData *WID, WINDSGeneralData *WGD)
{
  int num_sites = available_sensor_id.size();

  //std::vector<float> u_prof_1d, v_prof_1d;
  std::vector<float> wm, wms, u0_int, v0_int;
  std::vector<float> site_xcoord, site_ycoord, sum_wm, sum_wu, sum_wv;
  std::vector<float> dxx, dyy, u12, u34, v12, v34;
  std::vector<int> iwork, jwork;
  std::vector<int> k_mod;
  std::vector<int> surf_layer_height;

  //u_prof_1d.resize(num_sites * WGD->nz, 0.0);
  //v_prof_1d.resize(num_sites * WGD->nz, 0.0);
  wm.resize(num_sites * WGD->nx * WGD->ny, 0.0);
  wms.resize(num_sites * WGD->nx * WGD->ny, 0.0);
  u0_int.resize(num_sites * WGD->nz, 0.0);
  v0_int.resize(num_sites * WGD->nz, 0.0);
  sum_wm.resize(WGD->nx * WGD->ny, 0.0);
  sum_wu.resize(WGD->nx * WGD->ny * WGD->nz, 0.0);
  sum_wv.resize(WGD->nx * WGD->ny * WGD->nz, 0.0);
  site_xcoord.resize(num_sites, 0.0);
  site_ycoord.resize(num_sites, 0.0);
  iwork.resize(num_sites, 0);
  jwork.resize(num_sites, 0);
  dxx.resize(num_sites, 0.0);
  dyy.resize(num_sites, 0.0);
  u12.resize(num_sites * WGD->nz, 0.0);
  u34.resize(num_sites * WGD->nz, 0.0);
  v12.resize(num_sites * WGD->nz, 0.0);
  v34.resize(num_sites * WGD->nz, 0.0);
  k_mod.resize(WGD->nx * WGD->ny, 1);
  surf_layer_height.resize(WGD->nx * WGD->ny, 0);

  /*
    for (auto i = 0; i < num_sites; i++) {
    for (auto k = 0; k < WGD->nz; k++) {
      int id = k + i * WGD->nz;
      u_prof_1d[id] = u_prof[i][k];
      v_prof_1d[id] = v_prof[i][k];
    }
    }
  */

  for (auto i = 0; i < num_sites; i++) {
    site_xcoord[i] = WID->metParams->sensors[available_sensor_id[i]]->site_xcoord;
    site_ycoord[i] = WID->metParams->sensors[available_sensor_id[i]]->site_ycoord;
  }


  std::vector<float> x, y;
  x.resize(WGD->nx);
  for (int i = 0; i < WGD->nx; i++) {
    x[i] = (i - 0.5) * WGD->dx; /**< Location of face centers in x-dir */
  }

  y.resize(WGD->ny);
  for (auto j = 0; j < WGD->ny; j++) {
    y[j] = (j - 0.5) * WGD->dy; /**< Location of face centers in y-dir */
  }

  float *d_u_prof, *d_v_prof, *d_wm, *d_wms, *d_u0_int, *d_v0_int;
  float *d_x, *d_y, *d_site_xcoord, *d_site_ycoord, *d_sum_wm, *d_sum_wu, *d_sum_wv;
  float *d_u0, *d_v0, *d_w0;
  float *d_dxx, *d_dyy, *d_u12, *d_u34, *d_v12, *d_v34;
  int *d_iwork, *d_jwork, *d_site_id;
  int *d_terrain_face_id, *d_k_mod;
  float *d_z, *d_surf_layer_height;
  int *d_abl_height;

  hipMalloc((void **)&d_u_prof, num_sites * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_v_prof, num_sites * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_wm, num_sites * WGD->nx * WGD->ny * sizeof(float));
  hipMalloc((void **)&d_wms, num_sites * WGD->nx * WGD->ny * sizeof(float));
  hipMalloc((void **)&d_u0_int, num_sites * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_v0_int, num_sites * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_sum_wm, WGD->nx * WGD->ny * sizeof(float));
  hipMalloc((void **)&d_sum_wu, WGD->nx * WGD->ny * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_sum_wv, WGD->nx * WGD->ny * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_site_xcoord, num_sites * sizeof(float));
  hipMalloc((void **)&d_site_ycoord, num_sites * sizeof(float));
  hipMalloc((void **)&d_x, WGD->nx * sizeof(float));
  hipMalloc((void **)&d_y, WGD->ny * sizeof(float));
  hipMalloc((void **)&d_z, (WGD->nz - 1) * sizeof(float));
  hipMalloc((void **)&d_u0, WGD->numcell_face * sizeof(float));
  hipMalloc((void **)&d_v0, WGD->numcell_face * sizeof(float));
  hipMalloc((void **)&d_w0, WGD->numcell_face * sizeof(float));
  hipMalloc((void **)&d_iwork, num_sites * sizeof(int));
  hipMalloc((void **)&d_jwork, num_sites * sizeof(int));
  hipMalloc((void **)&d_site_id, num_sites * sizeof(int));
  hipMalloc((void **)&d_dxx, num_sites * sizeof(float));
  hipMalloc((void **)&d_dyy, num_sites * sizeof(float));
  hipMalloc((void **)&d_u12, num_sites * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_u34, num_sites * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_v12, num_sites * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_v34, num_sites * WGD->nz * sizeof(float));
  hipMalloc((void **)&d_terrain_face_id, WGD->nx * WGD->ny * sizeof(int));
  hipMalloc((void **)&d_k_mod, WGD->nx * WGD->ny * sizeof(int));
  hipMalloc((void **)&d_surf_layer_height, WGD->nx * WGD->ny * sizeof(float));
  hipMalloc((void **)&d_abl_height, num_sites * sizeof(int));

  hipMemcpy(d_u_prof, u_prof.data(), num_sites * WGD->nz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_v_prof, v_prof.data(), num_sites * WGD->nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_wm, wm.data(), num_sites * WGD->nx * WGD->ny * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_wms, wms.data(), num_sites * WGD->nx * WGD->ny * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_u0_int, u0_int.data(), num_sites * WGD->nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v0_int, v0_int.data(), num_sites * WGD->nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_wm, sum_wm.data(), WGD->nx * WGD->ny * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_wu, sum_wu.data(), WGD->nx * WGD->ny * WGD->nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_wv, sum_wv.data(), WGD->nx * WGD->ny * WGD->nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_site_xcoord, site_xcoord.data(), num_sites * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_site_ycoord, site_ycoord.data(), num_sites * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x.data(), WGD->nx * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y.data(), WGD->ny * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, WGD->z.data(), (WGD->nz - 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_u0, WGD->u0.data(), WGD->numcell_face * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v0, WGD->v0.data(), WGD->numcell_face * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w0, WGD->w0.data(), WGD->numcell_face * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_iwork, iwork.data(), num_sites * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_jwork, jwork.data(), num_sites * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_site_id, site_id.data(), num_sites * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dxx, dxx.data(), num_sites * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dyy, dyy.data(), num_sites * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_u12, u12.data(), num_sites * WGD->nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_u34, u34.data(), num_sites * WGD->nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v12, v12.data(), num_sites * WGD->nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v34, v34.data(), num_sites * WGD->nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_terrain_face_id, WGD->terrain_face_id.data(), WGD->nx * WGD->ny * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_k_mod, k_mod.data(), WGD->nx * WGD->ny * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_surf_layer_height, surf_layer_height.data(), WGD->nx * WGD->ny * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_abl_height, abl_height.data(), num_sites * sizeof(int), hipMemcpyHostToDevice);
	   
  BarnesScheme<<<1, 1>>>(d_u_prof, d_v_prof, d_wm, d_wms, d_u0_int, d_v0_int, d_x, d_y, d_z, d_site_xcoord, d_site_ycoord, d_sum_wm, d_sum_wu, d_sum_wv, d_u0, d_v0, d_w0, d_iwork, d_jwork, d_site_id, d_terrain_face_id, d_k_mod, d_dxx, d_dyy, d_u12, d_u34, d_v12, d_v34, num_sites, WGD->nx, WGD->ny, WGD->nz, WGD->dx, WGD->dy, asl_percent, d_abl_height, d_surf_layer_height);
  //cudaCheck(hipGetLastError());

  hipMemcpy(WGD->u0.data(), d_u0, WGD->numcell_face * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(WGD->v0.data(), d_v0, WGD->numcell_face * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(WGD->w0.data(), d_w0, WGD->numcell_face * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_u_prof);
  hipFree(d_v_prof);
  hipFree(d_wm);
  hipFree(d_wms);
  hipFree(d_u0_int);
  hipFree(d_v0_int);
  hipFree(d_site_xcoord);
  hipFree(d_site_ycoord);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_u0);
  hipFree(d_v0);
  hipFree(d_w0);
  hipFree(d_sum_wm);
  hipFree(d_sum_wu);
  hipFree(d_sum_wv);
  hipFree(d_iwork);
  hipFree(d_jwork);
  hipFree(d_site_id);
  hipFree(d_dxx);
  hipFree(d_dyy);
  hipFree(d_u12);
  hipFree(d_u34);
  hipFree(d_v12);
  hipFree(d_v34);
  hipFree(d_terrain_face_id);
  hipFree(d_k_mod);
  hipFree(d_surf_layer_height);
  hipFree(d_abl_height);
}
