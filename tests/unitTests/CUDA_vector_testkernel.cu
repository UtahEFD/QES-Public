#include "hip/hip_runtime.h"
#include "CUDA_vector_testkernel.h"
#include "util/VectorMath_CUDA.cuh"


__global__ void testCUDA_vectormath()
{
  // int id = (blockDim.x * blockIdx.x) + threadIdx.x;

  vec3 x, y, z;
  x = { 1.0f, 2.0f, 3.0f };
  y = { 1.0f, 2.0f, 3.0f };

  vec3 n = { 1.0f, 0.0f, 0.0f };

  z._1 = 3.0f * x._1 + y._1;
  z._2 = 3.0f * x._2 + y._2;
  z._3 = 3.0f * x._3 + y._3;

  float l = length(z);

  float s = dot(x, y);

  reflect(n, z);
}

__global__ void testCUDA_matmult(int length, mat3 *d_A, vec3 *d_b, vec3 *d_x)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int it = index; it < length; it += stride) {
    bool tt = invert(d_A[it]);
    multiply(d_A[it], d_b[it], d_x[it]);
  }
  return;
}

__global__ void testCUDA_invar(int length, mat3sym *d_tau, vec3 *d_invar)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int it = index; it < length; it += stride) {
    makeRealizable(10e-4, d_tau[it]);
    calcInvariants(d_tau[it], d_invar[it]);
  }
  return;
}

void test_gpu(const int &length)
{

  int gpuID = 0;
  hipError_t errorCheck = hipGetDevice(&gpuID);

  int blockCount = 1;
  hipDeviceGetAttribute(&blockCount, hipDeviceAttributeMultiprocessorCount, gpuID);
  std::cout << blockCount << std::endl;

  int threadsPerBlock = 32;
  hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, gpuID);
  std::cout << threadsPerBlock << std::endl;

  int blockSize = 1024;
  dim3 numberOfThreadsPerBlock(blockSize, 1, 1);
  dim3 numberOfBlocks(ceil(length / (float)(blockSize)), 1, 1);

  mat3 tmp = { 1, 2, 3, 2, 1, 2, 3, 2, 1 };
  std::vector<mat3> A;
  A.resize(length, tmp);

  std::vector<vec3> b;
  b.resize(length, { 1.0, 1.0, 1.0 });

  std::vector<vec3> x;
  x.resize(length, { 0.0, 0.0, 0.0 });

  std::vector<mat3sym> tau;
  // tau.resize(length, { 1, 2, 3, 1, 2, 1 });
  tau.resize(length, { 1, 0, 3, 0, 0, 1 });
  std::vector<vec3> invar;
  invar.resize(length, { 0.0, 0.0, 0.0 });

  if (errorCheck == hipSuccess) {
    // temp

    mat3 *d_A;
    hipMalloc((void **)&d_A, 9 * length * sizeof(float));
    vec3 *d_b;
    hipMalloc((void **)&d_b, 3 * length * sizeof(float));
    vec3 *d_x;
    hipMalloc((void **)&d_x, length * sizeof(vec3));

    mat3sym *d_tau;
    hipMalloc((void **)&d_tau, length * sizeof(mat3sym));
    vec3 *d_invar;
    hipMalloc((void **)&d_invar, length * sizeof(vec3));


    auto gpuStartTime = std::chrono::high_resolution_clock::now();

    // copy to the device
    hipMemcpy(d_A, A.data(), length * sizeof(mat3), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), length * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_tau, tau.data(), length * sizeof(mat3sym), hipMemcpyHostToDevice);

    // call kernel
    auto kernelStartTime = std::chrono::high_resolution_clock::now();
    testCUDA_vectormath<<<numberOfBlocks, numberOfThreadsPerBlock>>>();
    testCUDA_matmult<<<numberOfBlocks, numberOfThreadsPerBlock>>>(length, d_A, d_b, d_x);
    testCUDA_invar<<<numberOfBlocks, numberOfThreadsPerBlock>>>(length, d_tau, d_invar);
    // testCUDA_advection<<<numberOfBlocks, numberOfThreadsPerBlock>>>(length, d_x);
    hipDeviceSynchronize();
    auto kernelEndTime = std::chrono::high_resolution_clock::now();

    // cudamemcpy back to host
    hipMemcpy(A.data(), d_A, length * sizeof(mat3), hipMemcpyDeviceToHost);
    hipMemcpy(x.data(), d_x, length * sizeof(vec3), hipMemcpyDeviceToHost);
    hipMemcpy(tau.data(), d_tau, length * sizeof(mat3sym), hipMemcpyDeviceToHost);
    hipMemcpy(invar.data(), d_invar, length * sizeof(vec3), hipMemcpyDeviceToHost);

    auto gpuEndTime = std::chrono::high_resolution_clock::now();

    // cudafree
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);

    hipFree(d_tau);
    hipFree(d_invar);

    std::chrono::duration<double> kernelElapsed = kernelEndTime - kernelStartTime;
    std::cout << "kernel  elapsed time: " << kernelElapsed.count() << " s\n";
    std::chrono::duration<double> gpuElapsed = gpuEndTime - gpuStartTime;
    std::cout << "GPU  elapsed time: " << gpuElapsed.count() << " s\n";

    std::cout << "--------------------------------------" << std::endl;
    std::cout << "Sample of calculations" << std::endl;

    std::cout << A[0]._11 << " " << A[0]._12 << " " << A[0]._13 << std::endl;
    std::cout << A[0]._21 << " " << A[0]._22 << " " << A[0]._23 << std::endl;
    std::cout << A[0]._31 << " " << A[0]._32 << " " << A[0]._33 << std::endl;

    std::cout << std::endl;

    std::cout << x[0]._1 << " " << x[0]._2 << " " << x[0]._3 << std::endl;

    std::cout << std::endl;

    std::cout << tau[0]._11 << " " << tau[0]._12 << " " << tau[0]._13 << std::endl;
    std::cout << tau[0]._12 << " " << tau[0]._22 << " " << tau[0]._23 << std::endl;
    std::cout << tau[0]._13 << " " << tau[0]._23 << " " << tau[0]._33 << std::endl;

    std::cout << std::endl;

    std::cout << invar[0]._1 << " " << invar[0]._2 << " " << invar[0]._3 << std::endl;
    std::cout << "--------------------------------------" << std::endl;

  } else {
    printf("CUDA ERROR!\n");
  }
}
