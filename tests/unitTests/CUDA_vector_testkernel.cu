#include "hip/hip_runtime.h"
#include "CUDA_vector_testkernel.h"
#include "util/VectorMath_CUDA.cuh"


__global__ void testCUDA_vectormath()
{
  // int id = (blockDim.x * blockIdx.x) + threadIdx.x;

  vec3 x, y, z;
  x = { 1.0f, 2.0f, 3.0f };
  y = { 1.0f, 2.0f, 3.0f };

  vec3 n = { 1.0f, 0.0f, 0.0f };

  z._1 = 3.0f * x._1 + y._1;
  z._2 = 3.0f * x._2 + y._2;
  z._3 = 3.0f * x._3 + y._3;

  float l = length(z);

  float s = dot(x, y);

  reflect(n, z);
}

__global__ void testCUDA_multiply(int length, mat3 *d_A, vec3 *d_b, vec3 *d_x)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int it = index; it < length; it += stride) {
    multiply(d_A[it], d_b[it], d_x[it]);
  }
}

__global__ void testCUDA_invert(int length, mat3 *d_A)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int it = index; it < length; it += stride) {
    bool tt = invert(d_A[it]);
  }
}

__global__ void testCUDA_invariant(int length, mat3sym *d_tau, vec3 *d_invar)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int it = index; it < length; it += stride) {
    calcInvariants(d_tau[it], d_invar[it]);
  }
}

__global__ void testCUDA_realizable(int length, mat3sym *d_tau, vec3 *d_invar)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int it = index; it < length; it += stride) {
    makeRealizable(10e-4, d_tau[it]);
  }
}

void test_matrix_multiplication_gpu(const int &length, std::vector<mat3> &A, std::vector<vec3> &b, std::vector<vec3> &x)
{

  int gpuID = 0;
  hipError_t errorCheck = hipGetDevice(&gpuID);

  int blockCount = 1;
  hipDeviceGetAttribute(&blockCount, hipDeviceAttributeMultiprocessorCount, gpuID);
  // std::cout << blockCount << std::endl;

  int threadsPerBlock = 32;
  hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, gpuID);
  // std::cout << threadsPerBlock << std::endl;

  int blockSize = 1024;
  dim3 numberOfThreadsPerBlock(blockSize, 1, 1);
  dim3 numberOfBlocks(ceil(length / (float)(blockSize)), 1, 1);

  if (errorCheck == hipSuccess) {
    // temp

    mat3 *d_A;
    hipMalloc((void **)&d_A, 9 * length * sizeof(float));
    vec3 *d_b;
    hipMalloc((void **)&d_b, 3 * length * sizeof(float));
    vec3 *d_x;
    hipMalloc((void **)&d_x, length * sizeof(vec3));

    auto gpuStartTime = std::chrono::high_resolution_clock::now();

    // copy to the device
    hipMemcpy(d_A, A.data(), length * sizeof(mat3), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), length * sizeof(vec3), hipMemcpyHostToDevice);

    // call kernel
    auto kernelStartTime = std::chrono::high_resolution_clock::now();
    // testCUDA_vectormath<<<numberOfBlocks, numberOfThreadsPerBlock>>>();
    testCUDA_multiply<<<numberOfBlocks, numberOfThreadsPerBlock>>>(length, d_A, d_b, d_x);
    // testCUDA_invert<<<numberOfBlocks, numberOfThreadsPerBlock>>>(length, d_A);
    // testCUDA_invariant<<<numberOfBlocks, numberOfThreadsPerBlock>>>(length, d_tau, d_invar);
    //  testCUDA_advection<<<numberOfBlocks, numberOfThreadsPerBlock>>>(length, d_x);
    hipDeviceSynchronize();
    auto kernelEndTime = std::chrono::high_resolution_clock::now();

    // cudamemcpy back to host
    // hipMemcpy(A.data(), d_A, length * sizeof(mat3), hipMemcpyDeviceToHost);
    hipMemcpy(x.data(), d_x, length * sizeof(vec3), hipMemcpyDeviceToHost);

    auto gpuEndTime = std::chrono::high_resolution_clock::now();

    // cudafree
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);

    std::chrono::duration<double> kernelElapsed = kernelEndTime - kernelStartTime;
    std::cout << "kernel  elapsed time: " << kernelElapsed.count() << " s\n";
    std::chrono::duration<double> gpuElapsed = gpuEndTime - gpuStartTime;
    std::cout << "GPU  elapsed time: " << gpuElapsed.count() << " s\n";

  } else {
    printf("CUDA ERROR!\n");
  }
}


void test_matrix_inversion_gpu(const int &length, std::vector<mat3> &A)
{

  int gpuID = 0;
  hipError_t errorCheck = hipGetDevice(&gpuID);

  int blockCount = 1;
  hipDeviceGetAttribute(&blockCount, hipDeviceAttributeMultiprocessorCount, gpuID);
  // std::cout << blockCount << std::endl;

  int threadsPerBlock = 32;
  hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, gpuID);
  // std::cout << threadsPerBlock << std::endl;

  int blockSize = 1024;
  dim3 numberOfThreadsPerBlock(blockSize, 1, 1);
  dim3 numberOfBlocks(ceil(length / (float)(blockSize)), 1, 1);

  if (errorCheck == hipSuccess) {
    // temp

    mat3 *d_A;
    hipMalloc((void **)&d_A, 9 * length * sizeof(float));

    auto gpuStartTime = std::chrono::high_resolution_clock::now();

    // copy to the device
    hipMemcpy(d_A, A.data(), length * sizeof(mat3), hipMemcpyHostToDevice);

    // call kernel
    auto kernelStartTime = std::chrono::high_resolution_clock::now();
    testCUDA_invert<<<numberOfBlocks, numberOfThreadsPerBlock>>>(length, d_A);
    hipDeviceSynchronize();
    auto kernelEndTime = std::chrono::high_resolution_clock::now();

    // cudamemcpy back to host
    hipMemcpy(A.data(), d_A, length * sizeof(mat3), hipMemcpyDeviceToHost);

    auto gpuEndTime = std::chrono::high_resolution_clock::now();

    // cudafree
    hipFree(d_A);

    std::chrono::duration<double> kernelElapsed = kernelEndTime - kernelStartTime;
    std::cout << "kernel  elapsed time: " << kernelElapsed.count() << " s\n";
    std::chrono::duration<double> gpuElapsed = gpuEndTime - gpuStartTime;
    std::cout << "GPU  elapsed time: " << gpuElapsed.count() << " s\n";

  } else {
    printf("CUDA ERROR!\n");
  }
}

void test_matrix_invariants_gpu(const int &length, std::vector<mat3sym> &A, std::vector<vec3> &x)
{

  int gpuID = 0;
  hipError_t errorCheck = hipGetDevice(&gpuID);

  int blockCount = 1;
  hipDeviceGetAttribute(&blockCount, hipDeviceAttributeMultiprocessorCount, gpuID);
  // std::cout << blockCount << std::endl;

  int threadsPerBlock = 32;
  hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, gpuID);
  // std::cout << threadsPerBlock << std::endl;

  int blockSize = 1024;
  dim3 numberOfThreadsPerBlock(blockSize, 1, 1);
  dim3 numberOfBlocks(ceil(length / (float)(blockSize)), 1, 1);

  if (errorCheck == hipSuccess) {
    // temp

    mat3sym *d_A;
    hipMalloc((void **)&d_A, length * sizeof(mat3sym));
    vec3 *d_x;
    hipMalloc((void **)&d_x, length * sizeof(vec3));

    auto gpuStartTime = std::chrono::high_resolution_clock::now();

    // copy to the device
    hipMemcpy(d_A, A.data(), length * sizeof(mat3sym), hipMemcpyHostToDevice);

    // call kernel
    auto kernelStartTime = std::chrono::high_resolution_clock::now();
    testCUDA_invariant<<<numberOfBlocks, numberOfThreadsPerBlock>>>(length, d_A, d_x);
    hipDeviceSynchronize();
    auto kernelEndTime = std::chrono::high_resolution_clock::now();

    // cudamemcpy back to host
    hipMemcpy(x.data(), d_x, length * sizeof(vec3), hipMemcpyDeviceToHost);

    auto gpuEndTime = std::chrono::high_resolution_clock::now();

    // cudafree
    hipFree(d_A);
    hipFree(d_x);

    std::chrono::duration<double> kernelElapsed = kernelEndTime - kernelStartTime;
    std::cout << "kernel  elapsed time: " << kernelElapsed.count() << " s\n";
    std::chrono::duration<double> gpuElapsed = gpuEndTime - gpuStartTime;
    std::cout << "GPU  elapsed time: " << gpuElapsed.count() << " s\n";

  } else {
    printf("CUDA ERROR!\n");
  }
}
