/**
* Author: Andrew Larson <lars2865@d.umn.edu>
* Reason: CUDA kernel to find the min of a matrix - common functionality.
*         Used in urbViewer as part of QUICurbCUDA.
* Remark: Needs to be extended to handle more elements.
*/

#ifndef MIN_H
#define MIN_H

#include <stdio.h>

extern "C" void showError(char* loc);
extern "C" void cudaZero(float* d_abse, size_t nz, float value);

namespace QUIC 
{

	__global__ void k_min(float* d_array, float* d_mins) 
	{
		extern __shared__ float data[];

		int section = blockDim.x;
		int chunk   = section * 4;

		// Address Data
		float* s_1 = (float*)&data[0 * section];
		float* s_2 = (float*)&data[1 * section];
		float* s_3 = (float*)&data[2 * section];
		float* s_4 = (float*)&data[3 * section];
	
		int tidx = threadIdx.x;
		int s1I = blockIdx.x * chunk + 0 * section + tidx;
		int s2I = blockIdx.x * chunk + 1 * section + tidx;
		int s3I = blockIdx.x * chunk + 2 * section + tidx;
		int s4I = blockIdx.x * chunk + 3 * section + tidx;

		// Load Data
		s_1[tidx] = d_array[s1I];
		s_2[tidx] = d_array[s2I];
		s_3[tidx] = d_array[s3I];
		s_4[tidx] = d_array[s4I];

		__syncthreads();

		float min;

		while(section >= 1) 
		{			
			if(tidx < section) 
			{
				//Mini Reduce
				min = s_1[tidx];
				if(s_2[tidx] < min) {min = s_2[tidx];}
				if(s_3[tidx] < min) {min = s_3[tidx];}
				if(s_4[tidx] < min) {min = s_4[tidx];}
				s_1[tidx] = min;
			}

			section >>= 2;

			// Change where the section pointers are.
			s_1 = (float*)&data[0 * section];
			s_2 = (float*)&data[1 * section];
			s_3 = (float*)&data[2 * section];
			s_4 = (float*)&data[3 * section];

			__syncthreads();
		}

		if(tidx == 0) {d_mins[blockIdx.x] = s_1[0];}
	}

	__global__ void k_find_simple_min(float* d_array, int size, float* d_min) 
	{
		float cur_min = d_array[0];
		for(int i = 0; i < size; i++) 
		{
			if(d_array[i] < cur_min) {cur_min = d_array[i];}
		}
		*d_min = cur_min;
	}

	extern "C"
	void cudaMin(float* d_array, int size, float* d_min) 
	{
		// \\ todo same as max.cu -->
		int min_blocks = /*?*/ 65535; /*?*/ //From device info...
		// Gives a total of 65535 * 1024 sized array that min can be found of.

		int threads = 256; 
		int chunk = threads*4; //Must be power of 4 => threads a power of 4.		
		// Find largest power of 4 size
		int blocks     = int( size / chunk);
		int left_overs = size - blocks*chunk;
		int sharedMem  = chunk*sizeof(float);

		// Make sure the size if doable...
		if(size > min_blocks*chunk) 
		{
			printf
			(
				"cudaMin cannot min arrays larger than %d elements.", 
				min_blocks * chunk
			);
			return;
		}

		float* d_mins; hipMalloc((void**) &d_mins, (blocks + 1) * sizeof(float));
		cudaZero(d_mins, blocks + 1, 0.f);

		if(blocks > 0) 
		{
			k_min<<< dim3(blocks), dim3(threads), sharedMem >>>(d_array, d_mins); 
			showError("Min");
		}
		
		//Reduce the blocks if enough. 
		if(blocks >= chunk) 
		{
			cudaMin(d_mins, blocks, &d_mins[0]);
			blocks = 1;
		}

		if(left_overs) // Then eat them!!
		{
			k_find_simple_min<<< dim3(1), dim3(1) >>>
			(
				&d_array[size - left_overs], 
				left_overs, 
				&d_mins[blocks]
			); 
			showError("Simple Min (left_overs)");
		}

		k_find_simple_min<<< dim3(1), dim3(1) >>>
		(
			d_mins, 
			blocks + 1, 
			&d_mins[0]
		); 
		showError("Simple Min");

		hipMemcpy(d_min, &d_mins[0], sizeof(float), hipMemcpyDeviceToDevice);		
		hipFree(d_mins);
	}
}

#endif
